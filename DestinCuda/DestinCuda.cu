#include "hip/hip_runtime.h"
#include "AdviceData.h"
#include "DestinData.h"

#include <iostream>
#include <stdio.h>
#include <vector>
#include <fstream>
#include <ctime>
#include <sstream>
#include <sys/stat.h>
#include <math.h>

#ifdef _WIN32
#include <direct.h>
#else
// Linux only requirements...
#include <errno.h>
#include <unistd.h>
#include <string.h>
#include <stdlib.h>
#endif

using namespace std;

void PrintHelp()
{
    // ***************************
    // Print out how to use DeSTIN
    // ***************************

    cout << "Usage: DestinCuda CodeWord MAXCNT LayerToShow ParamsFile TrainingDataFile DestinOutputFile TargetDirectory [OutputDistillationLevel]" << endl;
    cout << "Where:" << endl;
    cout << "    CodeWord is an 11-digit value of the form." << endl;
    cout << "        RRRRXXYYYYY where RRRR is reserved, XX is number of classes, YYYYY is number of examples" << endl;
    cout << "    MAXCNT is the number of digits we show it to train the unsupervised DeSTIN architecture" << endl;
    cout << "    LayerToShow = layer written to output file; it is given as S:E:O:P:T where " << endl;
    cout << "        S = first layer to write" << endl;
    cout << "        E = last layer to write" << endl;
    cout << "        O = offset for movements to write" << endl;
    cout << "        P = period of movements to write" << endl;
    cout << "        T = type.  Nothing (and no !) is beliefs.  Type can be: " << endl;
    cout << "            A is belief in advice states computed by tabular method." << endl;
    cout << "            N is belief in advice states computed by neural network function approximator." << endl;
    cout << "            L is belief in advice states computed by linear function approximator." << endl;
    cout << "    ParamsFile is a file that has the run parameters" << endl;
    cout << "    TrainingDataFile is the binary data file for training.  A testing file with the SAME NAME and appended with _TESTING is assumed" << endl;
    cout << "    DestinOutputFile is the name of the DeSTIN network output file for saving." << endl;
    cout << "         Use -D as default, which is the experiment number with a .dat at the end, in the ../DiagnosticData directory" << endl;
    cout << "    TargetDirectory is where we want to put the MAIN OUTPUT DATA FILES.  We ALWAYS write an experiment marker to the " << endl;
    cout << "        ../DiagnosticData area.  But if you are writing out a lot of data you can specify another directory." << endl;
    cout << "        Put D for default which is the ../DiagnosticData area." << endl;
    cout << "    [OutputDistillationLevel] is optional.  If this exists it must be a number and currently its got to be 0.  "<<endl;
    cout << "        0 = regular outputs with a lot of details about movements and processing: this is our input to SampleAndStack"<<endl;
    cout << "        1 = outputs compatible with the regular distilled output of SampleAndStack. If you use this you can skip SampleAndStack.exe" << endl;
    cout << endl;
    cout << "-OR-" << endl;
    cout << endl;
    cout << "Usage: DestinCuda -F InputNetworkFile LayerToShow ParamsFile TrainingDataFile DestinOutputFile TargetDirectory [OutputDistillationLevel]" << endl;
    cout << "Where:" << endl;
    cout << "    -F signifies use a saved DeSTIN network file " << endl;
    cout << "    InputNetworkFile is the NAME of the saved DeSTIN network file" << endl;
    cout << "    All others are as in first usage type" << endl;
    cout << endl;
}

bool FileExists(string strFilename)
{
    // **************************
    // Does the given file exists
    // **************************
    // For detailed information look the return values of stat

    struct stat stFileInfo;
    bool blnReturn;
    int intStat;

    // Attempt to get the file attributes
    intStat = stat(strFilename.c_str(),&stFileInfo);
    if(intStat == 0) {
        // File exists
        blnReturn = true;
    }
    else
    {
        // File not exists or no permission
        blnReturn = false;
    }

    return(blnReturn);
}

string GetNextFileForDiagnostic()
{
    // *************************************
    // Find next available experimental file
    // *************************************
    // Check if there is a previous experiment inside ../DiagnosticData

    string strFileName;
    int iExperimentNumber=-1;
    bool bFileFound = true;
    while ( bFileFound )
    {
        iExperimentNumber++;
        stringstream buffer;
        buffer << "../DiagnosticData/DestinDiagnostics" << iExperimentNumber << ".csv";
        strFileName =  buffer.str();

        bFileFound = FileExists(strFileName);
    }

    return strFileName;
}

int MainDestinExperiments(int argc, char* argv[])
{
    // ********************************************
    // Main experiment of DeSTIN (Also called main)
    // ********************************************

    // File for diagnostic
    string strDiagnosticFileName;
    strDiagnosticFileName = GetNextFileForDiagnostic();

    // arguments processing

    // For debug information we output the command line to our Diagnostic file.
    string strCommandLineData = "";
    for( int i=0; i<argc; i++ )
    {
        strCommandLineData += argv[i];
        strCommandLineData += " ";
    }

    // Argument: DestinOutputFile or InputNetworkFile
    bool bCreateFromFile;
    string strDestinNetworkFileToRead;
    string strDestinNetworkFileToWrite;
    string FirstArg = argv[1];
    if ( FirstArg=="-F" )
    {
        // Argument: InputNetworkFile
        bCreateFromFile = true;
        strDestinNetworkFileToRead = argv[2];  // we read from this file...

        if ( !FileExists( strDestinNetworkFileToRead ) )
        {
            cout << "designated input network file named " << strDestinNetworkFileToRead.c_str() << " does not exist" << endl;
            return 0;
        }
    }
    else
    {
        // Argument: DestinOutputFile
        bCreateFromFile = false;
        strDestinNetworkFileToWrite = argv[6]; // we write to this file, and then we read from it too!!
        if ( strDestinNetworkFileToWrite == "-D" )
        {
            // If given -D
            strDestinNetworkFileToWrite=strDiagnosticFileName + "DestinNetwork.dat";
            cout << "Writing to default destin file name..." << endl;
        }
        strDestinNetworkFileToRead = strDestinNetworkFileToWrite;
    }

    // Argument: LayerToShow
    // Structure of processing S:E:O:P:T
    // List of default values
    int FirstLayerToShowHECK = 3;
    int LastLayerToShow = FirstLayerToShowHECK;
    int iMovementOutputOffset = 0;
    int iMovementOutputPeriod = 1;
    OutputTypes eTypeOfOutput = eBeliefs;

    string sLayerSpecs = argv[3];
        int iColon = sLayerSpecs.find(":");
        if ( iColon == -1 || sLayerSpecs.substr(iColon).empty() )  //first layer = last layer, and no sampling specified.
        {
            // S
            FirstLayerToShowHECK=atoi(sLayerSpecs.c_str());
            LastLayerToShow=FirstLayerToShowHECK;
        }
        else
        {
            // S:E
            FirstLayerToShowHECK=atoi(sLayerSpecs.substr(0,1).c_str());
            LastLayerToShow=atoi(sLayerSpecs.substr(iColon+1,1).c_str());
            sLayerSpecs = sLayerSpecs.substr(iColon+1);
            iColon = sLayerSpecs.find(":");
            if ( iColon!=-1 || !( sLayerSpecs.substr(iColon).empty() ) )
            {
                //S:E:O
                sLayerSpecs = sLayerSpecs.substr(iColon+1);
                iMovementOutputOffset = atoi(sLayerSpecs.substr(0,1).c_str());
                iColon = sLayerSpecs.find(":");
                if ( iColon!=-1 || !( sLayerSpecs.substr(iColon).empty() ) )
                {
                    //S:E:O:P
                    sLayerSpecs = sLayerSpecs.substr(iColon+1);
                    iMovementOutputPeriod = atoi(sLayerSpecs.substr(0,1).c_str());
                    iColon = sLayerSpecs.find(":");
                    if ( iColon!=-1 || !( sLayerSpecs.substr(iColon).empty() ) )
                    {
                        //S:E:O:P:T
                        sLayerSpecs = sLayerSpecs.substr(iColon+1);
                        if ( sLayerSpecs.substr(0,1)=="A" )
                        {
                            eTypeOfOutput = eBeliefInAdviceTabular;
                        }
                        else if ( sLayerSpecs.substr(0,1)=="B" )
                        {
                            eTypeOfOutput = eBeliefs;
                        }
                        else if ( sLayerSpecs.substr(0,1)=="N" )
                        {
                            eTypeOfOutput = eBeliefInAdviceNNFA;
                        }
                        else if ( sLayerSpecs.substr(0,1)=="L" )
                        {
                            eTypeOfOutput = eBeliefInAdviceLinearFA;
                        }
                        else
                        {
                            cout << "Do not understand the output type " << sLayerSpecs.c_str() << endl;
                            return 0;
                        }
                    }
                }
            }
        }

    // Argument: TargetDirectory
    // A given location instead or default
    string strDiagnosticFileNameForData;
    string strArg7 = argv[7];
    if ( strArg7 == "D" )
    {
        strDiagnosticFileNameForData = strDiagnosticFileName;
    }
    else
    {
        // Buffer with path + filename where to put diagnostic data
        stringstream buffer;
        buffer << strArg7.c_str() << "/" << strDiagnosticFileName;
        strDiagnosticFileNameForData = buffer.str();
    }

    // Optional argument: OutputDistillationLevel
    // This will write out a distilled movement log file this movement log matches that what SampleAndStack would produce.
    int OutputDistillationLevel = 0; //default level
    if ( argc == 9 )
    {
        OutputDistillationLevel = atoi(argv[8]);
    }

    // **********************
    // Loading data source(s)
    // **********************
    // Arguments: TrainingDataFile
    // Load the training file for DeSTIN
    string strDestinTrainingFileName = argv[5];

    // Data object containing source training
    DestinData DataSourceForTraining;

    int NumberOfUniqueLabels;
    DataSourceForTraining.LoadFile(strDestinTrainingFileName.c_str());
    NumberOfUniqueLabels = DataSourceForTraining.GetNumberOfUniqueLabels();
    if ( NumberOfUniqueLabels==0 )
    {
        cout << "There seems to be something off with data source " << strDestinTrainingFileName.c_str() << endl;
        return 0;
    }

    // A vector with all the labels of the data source
    vector<int> vLabelList;
    DataSourceForTraining.GetUniqueLabels(vLabelList);

    // Load the test file for DeSTIN
    string strTesting = strDestinTrainingFileName;
    strTesting = strTesting + "_TESTING";
    // Data object of test source
    DestinData DataSourceForTesting;

    DataSourceForTesting.LoadFile((char*)(strTesting.c_str()));
    if ( DataSourceForTesting.GetNumberOfUniqueLabels()!=NumberOfUniqueLabels )
    {
        cout << "Test set does not have the same number of labels as train set " << endl;
        return 0;
    }
    // end of data loading

    return 0;
}

int main(int argc, char* argv[])
{
    // ********************
    // Startup check DeSTIN
    // ********************

    if ( argc==8 || argc==9 )
    {
        return MainDestinExperiments(argc,argv);
    }
    else
    {
        PrintHelp();
        return 0;
    }
}
