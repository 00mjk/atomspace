#include "hip/hip_runtime.h"
#include "AdviceData.h"
#include "DestinData.h"

#include <iostream>
#include <stdio.h>
#include <vector>
#include <fstream>
#include <ctime>
#include <sstream>
#include <sys/stat.h>
#include <math.h>

#ifdef _WIN32
#include <direct.h>
#else
// Linux only requirements...
#include <errno.h>
#include <unistd.h>
#include <string.h>
#include <stdlib.h>
#endif

using namespace std;

void PrintHelp()
{
    // ***************************
    // Print out how to use DeSTIN
    // ***************************

    cout << "Usage: DestinCuda CodeWord MAXCNT LayerToShow ParamsFile TrainingDataFile DestinOutputFile TargetDirectory [OutputDistillationLevel]" << endl;
    cout << "Where:" << endl;
    cout << "    CodeWord must have 11 digits RRRRXXYYYYY" << endl;
    cout << "        RRRR  = 0000 to 9999 where 0000 is real random time." << endl;
    cout << "        XX    = 01 to 99 number of classes will not be higher then training file." << endl;
    cout << "        YYYYY = 00000 to 99999 number of examples of each class." << endl;
    cout << "                00000 means RANDOMLY PICK EXAMPLES until we finish clustering, period, up to max iterations." << endl;
    cout << "    MAXCNT is the number of digits we show it to train the unsupervised DeSTIN architecture" << endl;
    cout << "    LayerToShow = layer written to output file; it is given as S:E:O:P:T where " << endl;
    cout << "        S = first layer to write" << endl;
    cout << "        E = last layer to write" << endl;
    cout << "        O = offset for movements to write" << endl;
    cout << "        P = period of movements to write" << endl;
    cout << "        T = type.  Nothing (and no !) is beliefs.  Type can be: " << endl;
    cout << "            A is belief in advice states computed by tabular method." << endl;
    cout << "            N is belief in advice states computed by neural network function approximator." << endl;
    cout << "            L is belief in advice states computed by linear function approximator." << endl;
    cout << "    ParamsFile is a file that has the run parameters" << endl;
    cout << "    TrainingDataFile is the binary data file for training.  A testing file with the SAME NAME and appended with _TESTING is assumed" << endl;
    cout << "    DestinOutputFile is the name of the DeSTIN network output file for saving." << endl;
    cout << "         Use -D as default, which is the experiment number with a .dat at the end, in the ../DiagnosticData directory" << endl;
    cout << "    TargetDirectory is where we want to put the MAIN OUTPUT DATA FILES.  We ALWAYS write an experiment marker to the " << endl;
    cout << "        ../DiagnosticData area.  But if you are writing out a lot of data you can specify another directory." << endl;
    cout << "        Put D for default which is the ../DiagnosticData area." << endl;
    cout << "    [OutputDistillationLevel] is optional.  If this exists it must be a number and currently its got to be 0.  "<<endl;
    cout << "        0 = regular outputs with a lot of details about movements and processing: this is our input to SampleAndStack"<<endl;
    cout << "        1 = outputs compatible with the regular distilled output of SampleAndStack. If you use this you can skip SampleAndStack.exe" << endl;
    cout << endl;
    cout << "-OR-" << endl;
    cout << endl;
    cout << "Usage: DestinCuda -F InputNetworkFile LayerToShow ParamsFile TrainingDataFile DestinOutputFile TargetDirectory [OutputDistillationLevel]" << endl;
    cout << "Where:" << endl;
    cout << "    -F signifies use a saved DeSTIN network file " << endl;
    cout << "    InputNetworkFile is the NAME of the saved DeSTIN network file" << endl;
    cout << "    All others are as in first usage type" << endl;
    cout << endl;
}

bool FileExists(string strFilename)
{
    // **************************
    // Does the given file exists
    // **************************
    // For detailed information look the return values of stat

    struct stat stFileInfo;
    bool blnReturn;
    int intStat;

    // Attempt to get the file attributes
    intStat = stat(strFilename.c_str(),&stFileInfo);
    if(intStat == 0) {
        // File exists
        blnReturn = true;
    }
    else
    {
        // File not exists or no permission
        blnReturn = false;
    }

    return(blnReturn);
}

string GetNextFileForDiagnostic()
{
    // *************************************
    // Find next available experimental file
    // *************************************
    // Check if there is a previous experiment inside ../DiagnosticData

    string strFileName;
    int iExperimentNumber=-1;
    bool bFileFound = true;
    while ( bFileFound )
    {
        iExperimentNumber++;
        stringstream buffer;
        buffer << "../DiagnosticData/DestinDiagnostics" << iExperimentNumber << ".csv";
        strFileName =  buffer.str();

        bFileFound = FileExists(strFileName);
    }

    return strFileName;
}

int MainDestinExperiments(int argc, char* argv[])
{
    // ********************************************
    // Main experiment of DeSTIN (Also called main)
    // ********************************************

    // File for diagnostic
    string strDiagnosticFileName;
    strDiagnosticFileName = GetNextFileForDiagnostic();

    // arguments processing

    // For debug information we output the command line to our Diagnostic file.
    string strCommandLineData = "";
    for( int i=0; i<argc; i++ )
    {
        strCommandLineData += argv[i];
        strCommandLineData += " ";
    }

    // Argument: DestinOutputFile or InputNetworkFile
    bool bCreateFromFile;
    string strDestinNetworkFileToRead;
    string strDestinNetworkFileToWrite;
    string FirstArg = argv[1];
    if ( FirstArg=="-F" )
    {
        // Argument: InputNetworkFile
        bCreateFromFile = true;
        strDestinNetworkFileToRead = argv[2];  // we read from this file...

        if ( !FileExists( strDestinNetworkFileToRead ) )
        {
            cout << "designated input network file named " << strDestinNetworkFileToRead.c_str() << " does not exist" << endl;
            return 0;
        }
        cout << "Writing destin file to: " << strDestinNetworkFileToWrite << endl;
    }
    else
    {
        // Argument: DestinOutputFile
        bCreateFromFile = false;
        strDestinNetworkFileToWrite = argv[6]; // we write to this file, and then we read from it too!!
        if ( strDestinNetworkFileToWrite == "-D" )
        {
            // If given -D
            strDestinNetworkFileToWrite=strDiagnosticFileName + "DestinNetwork.dat";
            cout << "Writing default destin file to: " << strDestinNetworkFileToWrite << endl;
        }
        strDestinNetworkFileToRead = strDestinNetworkFileToWrite;
    }

    // Argument: LayerToShow
    // Structure of processing S:E:O:P:T
    // List of default values
    int FirstLayerToShowHECK = 3;
    int LastLayerToShow = FirstLayerToShowHECK;
    int iMovementOutputOffset = 0;
    int iMovementOutputPeriod = 1;
    OutputTypes eTypeOfOutput = eBeliefs;

    string sLayerSpecs = argv[3];
    int iColon = sLayerSpecs.find(":");
    if ( iColon == -1 || sLayerSpecs.substr(iColon).empty() )  //first layer = last layer, and no sampling specified.
    {
        // S
        FirstLayerToShowHECK=atoi(sLayerSpecs.c_str());
        LastLayerToShow=FirstLayerToShowHECK;
    }
    else
    {
        // S:E
        FirstLayerToShowHECK=atoi(sLayerSpecs.substr(0,1).c_str());
        LastLayerToShow=atoi(sLayerSpecs.substr(iColon+1,1).c_str());
        sLayerSpecs = sLayerSpecs.substr(iColon+1);
        iColon = sLayerSpecs.find(":");
        if ( iColon!=-1 && !( sLayerSpecs.substr(iColon).empty() ) )
        {
            //S:E:O
            sLayerSpecs = sLayerSpecs.substr(iColon+1);
            iMovementOutputOffset = atoi(sLayerSpecs.substr(0,1).c_str());
            iColon = sLayerSpecs.find(":");
            if ( iColon!=-1 && !( sLayerSpecs.substr(iColon).empty() ) )
            {
                //S:E:O:P
                sLayerSpecs = sLayerSpecs.substr(iColon+1);
                iMovementOutputPeriod = atoi(sLayerSpecs.substr(0,1).c_str());
                iColon = sLayerSpecs.find(":");
                if ( iColon!=-1 && !( sLayerSpecs.substr(iColon).empty() ) )
                {
                    //S:E:O:P:T
                    sLayerSpecs = sLayerSpecs.substr(iColon+1);
                    if ( sLayerSpecs.substr(0,1)=="A" )
                    {
                        eTypeOfOutput = eBeliefInAdviceTabular;
                    }
                    else if ( sLayerSpecs.substr(0,1)=="B" )
                    {
                        eTypeOfOutput = eBeliefs;
                    }
                    else if ( sLayerSpecs.substr(0,1)=="N" )
                    {
                        eTypeOfOutput = eBeliefInAdviceNNFA;
                    }
                    else if ( sLayerSpecs.substr(0,1)=="L" )
                    {
                        eTypeOfOutput = eBeliefInAdviceLinearFA;
                    }
                    else
                    {
                        cout << "Do not understand the output type " << sLayerSpecs.c_str() << endl;
                        return 0;
                    }
                }
            }
        }
    }

    // Argument: TargetDirectory
    // A given location instead or default
    string strDiagnosticFileNameForData;
    string strArg7 = argv[7];
    if ( strArg7 == "D" )
    {
        strDiagnosticFileNameForData = strDiagnosticFileName;
    }
    else
    {
        // Buffer with path + filename where to put diagnostic data
        stringstream buffer;
        buffer << strArg7.c_str() << "/" << strDiagnosticFileName;
        strDiagnosticFileNameForData = buffer.str();
    }

    // Optional argument: OutputDistillationLevel
    // This will write out a distilled movement log file this movement log matches that what SampleAndStack would produce.
    int OutputDistillationLevel = 0; //default level
    if ( argc == 9 )
    {
        OutputDistillationLevel = atoi(argv[8]);
    }

    // **********************
    // Loading data source(s)
    // **********************
    // Arguments: TrainingDataFile
    // Load the training file for DeSTIN
    string strDestinTrainingFileName = argv[5];

    // Data object containing source training
    DestinData DataSourceForTraining;

    int NumberOfUniqueLabels;
    DataSourceForTraining.LoadFile(strDestinTrainingFileName.c_str());
    NumberOfUniqueLabels = DataSourceForTraining.GetNumberOfUniqueLabels();
    if ( NumberOfUniqueLabels==0 )
    {
        cout << "There seems to be something off with data source " << strDestinTrainingFileName.c_str() << endl;
        return 0;
    }

    // A vector with all the labels of the data source
    vector<int> vLabelList;
    DataSourceForTraining.GetUniqueLabels(vLabelList);

    // Load the test file for DeSTIN
    string strTesting = strDestinTrainingFileName;
    strTesting = strTesting + "_TESTING";
    // Data object of test source
    DestinData DataSourceForTesting;

    DataSourceForTesting.LoadFile((char*)(strTesting.c_str()));
    if ( DataSourceForTesting.GetNumberOfUniqueLabels()!=NumberOfUniqueLabels )
    {
        cout << "Test set does not have the same number of labels as train set " << endl;
        return 0;
    }

    // **************************
    // Preparing working data set
    // **************************
    // now get the data set creation parameters
    int NumberOfUniqueLabelsToUse;
    int MAX_CNT = 1000;
    int iTestSequence = 0;
    string ParametersFileName;
    vector< pair<int,int> > vIndicesAndGTLabelToUse;

    if ( bCreateFromFile==false )
    {
        // Argument: MAXCNT
        MAX_CNT=atoi(argv[2]);
        // Argument: CodeWord
        iTestSequence=atoi(argv[1]);
        string sCodeWord=argv[1];
        if (sCodeWord.length() != 11 )
        {
            PrintHelp();
            return 0;
        }
        // First part of code word RRRR = for time seeding
        string sNumInp;
        sNumInp= sCodeWord.substr(0,4);

        // if the first 4 digits are 0000 make a TRUE random, otherwise use the complete number.
        int iReserve = atoi( sNumInp.c_str() );
        if ( iReserve == 0 )
        {
            srand( time(NULL) );
        }
        else
        {
            int iRandSeed = iTestSequence;
            srand( (unsigned int)iRandSeed );
        }

        // Second part of code word XX = number of inputs
        sNumInp = sCodeWord.substr(4,2);
        NumberOfUniqueLabelsToUse = atoi( sNumInp.c_str() );

        // Last part of code word YYYYY
        int iNumberOfExamplesFromEachLabel;
        sNumInp = sCodeWord.substr(6,5);
        iNumberOfExamplesFromEachLabel=atoi( sNumInp.c_str() );

        // if iNumberOfExamplesFromEachLabel is 0 we randomly pick examples from the available
        // classes and only show them ONE TIME
        // Generate the examples from the dictates given here.
        vector< pair<int,int> > LabelsAndIndicesForUse;
        cout << "------------------" << endl;
        int DestinTrainSampleStep = 1;
        if(iNumberOfExamplesFromEachLabel == 0)
        {
            DestinTrainSampleStep = 25;
        }
        for(int iLabel=0;iLabel<NumberOfUniqueLabelsToUse;iLabel++)
        {
            int cnt = 0;
            vector<int> IndicesForThisLabel;
            DataSourceForTraining.GetIndicesForThisLabel(iLabel,IndicesForThisLabel);
            if ( IndicesForThisLabel.size() > iNumberOfExamplesFromEachLabel && iNumberOfExamplesFromEachLabel != 0)
            {
                for(int jj=0;jj<iNumberOfExamplesFromEachLabel;jj++)
                {
                    cnt++;
                    pair<int,int> P;
                    P.first = IndicesForThisLabel[jj];
                    P.second = iLabel;
                    LabelsAndIndicesForUse.push_back(P);
                }
            }
            else
            {
                for(int jj=0;jj<IndicesForThisLabel.size();jj=jj+DestinTrainSampleStep)
                {
                    cnt++;
                    pair<int,int> P;
                    P.first = IndicesForThisLabel[jj];
                    P.second = iLabel;
                    LabelsAndIndicesForUse.push_back(P);
                }

            }
            cout << "Label: " << iLabel << " got " << cnt << " unique sample(s)." << endl;
        }
        iNumberOfExamplesFromEachLabel = LabelsAndIndicesForUse.size()/NumberOfUniqueLabelsToUse;

        // Now generate MAX_CNT+1000 random numbers from 0 to LabelsAndIndicesForUse-1
        // and use these to populate vIndicesAndGTLabelToUse

        // Debug list of labels to be used
        int Picked[NumberOfUniqueLabels];
        for(int jj=0;jj<NumberOfUniqueLabels;jj++)
        {
            Picked[jj]=0;
        }

        int Digit;
        int iChoice;
        for(int jj=0;jj<MAX_CNT;jj++)
        {
            //pick the digit first...
            Digit = rand() % NumberOfUniqueLabelsToUse;
            iChoice = Digit * iNumberOfExamplesFromEachLabel;
            iChoice = iChoice+rand() % iNumberOfExamplesFromEachLabel;

            pair<int,int> P;
            P = LabelsAndIndicesForUse[iChoice];

            vIndicesAndGTLabelToUse.push_back( P );
            // Debug counter of labels used by label
            Picked[P.second] += 1;
        }

        // Debug information on amount of examples we use each label
        cout << "------------------" << endl;
        for(int jj=0;jj<NumberOfUniqueLabels;jj++)
        {
            cout << "Label: " << jj << " will show " << Picked[jj] << " sample(s)." << endl;
        }
        cout << "------------------" << endl;
    }  //check on bCreateFromFile==false
    else
    {
        // TODO: We want to create the network from an INPUT FILE!
        cout << "We want to create the network from an INPUT FILE!" << endl;
    }

    // Argument: ParamsFile
    // A configuration file for DeSTIN
    ParametersFileName=argv[4];
    if ( !FileExists(ParametersFileName) )
    {
        if ( bCreateFromFile == false )
        {
            cout << "Parameters file name does not exist" << endl;
            return 0;
        }
        //otherwise we are OK, we don't need parameters if the network file was already created, as we are supposed to get the parameters from it...I think...
    }

    return 0;
}

int main(int argc, char* argv[])
{
    // ********************
    // Startup check DeSTIN
    // ********************
    // There should be 8 or 9 arguments at this time if not show how to use DeSTIN
    if ( argc==8 || argc==9 )
    {
        return MainDestinExperiments(argc,argv);
    }
    else
    {
        PrintHelp();
        return 0;
    }
}
