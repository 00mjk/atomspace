#include "hip/hip_runtime.h"
#include "DestinData.h"

#include <iostream>
#include <stdio.h>
#include <vector>
#include <fstream>
#include <ctime>
#include <sstream>
#include <sys/stat.h>
#include <math.h>

#ifdef _WIN32
#include <direct.h>
#else
// Linux only requirements...
#include <errno.h>
#include <unistd.h>
#include <string.h>
#include <stdlib.h>
#endif

using namespace std;

void PrintHelp()
{
    // ***************************
    // Print out how to use DeSTIN
    // ***************************

    cout << "Usage: DestinCuda CodeWord MAXCNT LayerToShow ParamsFile TrainingDataFile DestinOutputFile TargetDirectory [OutputDistillationLevel]" << endl;
    cout << "Where:" << endl;
    cout << "    CodeWord is an 11-digit value of the form." << endl;
    cout << "        RRRRXXYYYYY where RRRR is reserved, XX is number of classes, YYYYY is number of examples" << endl;
    cout << "    MAXCNT is the number of digits we show it to train the unsupervised DeSTIN architecture" << endl;
    cout << "    LayerToShow = layer written to output file; it is given as S:E:O:P:T where " << endl;
    cout << "        S = first layer to write" << endl;
    cout << "        E = last layer to write" << endl;
    cout << "        O = offset for movements to write" << endl;
    cout << "        P = period of movements to write" << endl;
    cout << "        T = type.  Nothing (and no !) is beliefs.  Type can be: " << endl;
    cout << "            A is belief in advice states computed by tabular method." << endl;
    cout << "            N is belief in advice states computed by neural network function approximator." << endl;
    cout << "            L is belief in advice states computed by linear function approximator." << endl;
    cout << "    ParamsFile is a file that has the run parameters" << endl;
    cout << "    TrainingDataFile is the binary data file for training.  A testing file with the SAME NAME and appended with _TESTING is assumed" << endl;
    cout << "    DestinOutputFile is the name of the DeSTIN network output file for saving." << endl;
    cout << "         Use -D as default, which is the experiment number with a .dat at the end, in the ../DiagnosticData directory" << endl;
    cout << "    TargetDirectory is where we want to put the MAIN OUTPUT DATA FILES.  We ALWAYS write an experiment marker to the " << endl;
    cout << "        ../DiagnosticData area.  But if you are writing out a lot of data you can specify another directory." << endl;
    cout << "        Put D for default which is the ../DiagnosticData area." << endl;
    cout << "    [OutputDistillationLevel] is optional.  If this exists it must be a number and currently its got to be 0.  "<<endl;
    cout << "        0 = regular outputs with a lot of details about movements and processing: this is our input to SampleAndStack"<<endl;
    cout << "        1 = outputs compatible with the regular distilled output of SampleAndStack. If you use this you can skip SampleAndStack.exe" << endl;
    cout << endl;
    cout << "-OR-" << endl;
    cout << endl;
    cout << "Usage: DestinCuda -F InputNetworkFile LayerToShow ParamsFile TrainingDataFile DestinOutputFile TargetDirectory [OutputDistillationLevel]" << endl;
    cout << "Where:" << endl;
    cout << "    -F signifies use a saved DeSTIN network file " << endl;
    cout << "    InputNetworkFile is the NAME of the saved DeSTIN network file" << endl;
    cout << "    All others are as in first usage type" << endl;
    cout << endl;
}

bool FileExists(string strFilename)
{
    // **************************
    // Does the given file exists
    // **************************
    // For detailed information look the return values of stat

    struct stat stFileInfo;
    bool blnReturn;
    int intStat;

    // Attempt to get the file attributes
    intStat = stat(strFilename.c_str(),&stFileInfo);
    if(intStat == 0) {
        // File exists
        blnReturn = true;
    }
    else
    {
        // File not exists or no permission
        blnReturn = false;
    }

    return(blnReturn);
}

string GetNextFileForDiagnostic()
{
    // *************************************
    // Find next available experimental file
    // *************************************
    // Check if there is a previous experiment inside ../DiagnosticData

    string strFileName;
    int iExperimentNumber=-1;
    bool bFileFound = true;
    while ( bFileFound )
    {
        iExperimentNumber++;
        stringstream buffer;
        buffer << "../DiagnosticData/DestinDiagnostics" << iExperimentNumber << ".csv";
        strFileName =  buffer.str();

        bFileFound = FileExists(strFileName);
    }

    return strFileName;
}

int MainDestinExperiments(int argc, char* argv[])
{
    // ********************************************
    // Main experiment of DeSTIN (Also called main)
    // ********************************************

    // File for diagnostic
    string strDiagnosticFileName;
    strDiagnosticFileName = GetNextFileForDiagnostic();

    // arguments processing

    // For debug information we output the command line to our Diagnostic file.
    string strCommandLineData = "";
    for( int i=0; i<argc; i++ )
    {
        strCommandLineData += argv[i];
        strCommandLineData += " ";
    }

    // Argument: DestinOutputFile or InputNetworkFile
    bool bCreateFromFile;
    string strDestinNetworkFileToRead;
    string strDestinNetworkFileToWrite;
    string FirstArg = argv[1];
    if ( FirstArg=="-F" )
    {
        // Argument: InputNetworkFile
        bCreateFromFile = true;
        strDestinNetworkFileToRead = argv[2];  // we read from this file...

        if ( !FileExists( strDestinNetworkFileToRead ) )
        {
            cout << "designated input network file named " << strDestinNetworkFileToRead.c_str() << " does not exist" << endl;
            return 0;
        }
    }
    else
    {
        // Argument: DestinOutputFile
        bCreateFromFile = false;
        strDestinNetworkFileToWrite = argv[6]; // we write to this file, and then we read from it too!!
        if ( strDestinNetworkFileToWrite == "-D" )
        {
            // If given -D
            strDestinNetworkFileToWrite=strDiagnosticFileName + "DestinNetwork.dat";
            cout << "Writing to default destin file name..." << endl;
        }
        strDestinNetworkFileToRead = strDestinNetworkFileToWrite;
    }

    // Argument: TargetDirectory
    // A given location instead or default
    string strDiagnosticFileNameForData;
    string strArg7 = argv[7];
    if ( strArg7 == "D" )
    {
        strDiagnosticFileNameForData = strDiagnosticFileName;
    }
    else
    {
        // Buffer with path + filename where to put diagnostic data
        stringstream buffer;
        buffer << strArg7.c_str() << "/" << strDiagnosticFileName;
        strDiagnosticFileNameForData = buffer.str();
    }

    // Optional argument: OutputDistillationLevel
    // This will write out a distilled movement log file this movement log matches that what SampleAndStack would produce.
    int OutputDistillationLevel = 0; //default level
    if ( argc == 9 )
    {
        OutputDistillationLevel = atoi(argv[8]);
    }



    return 0;
}

int main(int argc, char* argv[])
{
    // ********************
    // Startup check DeSTIN
    // ********************

    if ( argc==8 || argc==9 )
    {
        return MainDestinExperiments(argc,argv);
    }
    else
    {
        PrintHelp();
        return 0;
    }
}
