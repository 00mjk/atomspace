#include "hip/hip_runtime.h"
#include "DestinData.h"

#include <fstream>
#include <map>
#include <math.h>
#include <iostream>

using namespace std;

DestinData::DestinData(void)
{
	mLastImageIndex=-1;
	mRows=0;
	mCols=0;
	mShiftedImageCache=NULL;
}

DestinData::~DestinData(void)
{
	if ( mShiftedImageCache != NULL ) 
	{
		for(int r=0;r<mRows;r++)
		{
			delete mShiftedImageCache[r];
		}
		delete mShiftedImageCache;
	}
}

int DestinData::GetLabel(int iIndexOfImage)
{
	return mLabels[iIndexOfImage];
}


void DestinData::LoadFile(const char* sFileName)
{
	mImagePointer.clear();
	mLabels.clear();
	mUniqueLabels.clear();
	mMapLabelToIndexVector.clear();

	std::ifstream stmIn;
	stmIn.open(sFileName,ios::in | ios::binary );

	int iSignals;
	stmIn.read( (char*)&iSignals,sizeof(iSignals));
	stmIn.read( (char*)&mRows, sizeof(mRows) );
	stmIn.read( (char*)&mCols, sizeof(mCols) );

	int iLabel;
	unsigned char* cImageData;
	float** fImageDataByRow;
	float *fRow;

	map<int, vector<int> >::iterator it;

	for(int i=0; i<iSignals; i++)
	{
		stmIn.read( (char*)&iLabel, sizeof(iLabel) );
		mLabels.push_back(iLabel);
		it = mMapLabelToIndexVector.find(iLabel);
		if ( it==mMapLabelToIndexVector.end() ) //didn't find it
		{
			vector<int> vInt;
			vInt.push_back((int)(mLabels.size())-1);
			mMapLabelToIndexVector[iLabel]=vInt;
			mUniqueLabels.push_back(iLabel);
		}
		else
		{
			it->second.push_back((int)(mLabels.size())-1); //add the new vectors index...
		}
		cImageData = new unsigned char[mRows*mCols];
		stmIn.read( (char*)cImageData, mRows*mCols );  //The data is stored row 1, column 1-end, row 2, column 1-end, etc
		// so we have to transpose it here...
		fImageDataByRow = new float*[mRows];
		unsigned char* p = cImageData;
		for(int r=0;r<mRows;r++)
		{
			fRow = new float[mCols];
			for(int c=0;c<mCols;c++)
			{
				fRow[c]=((float)*p)/255.0;
				p++;
			}
			fImageDataByRow[r]=fRow;
		}
		delete cImageData;
		mImagePointer.push_back(fImageDataByRow);
	}
	stmIn.close();
	cout << "Finished reading file." << endl;
}

int DestinData::GetNumberOfImages()
{
	return mLabels.size();
}

int DestinData::GetNumberOfUniqueLabels()
{
	return mUniqueLabels.size();
}

void DestinData::GetLabelList(vector<int>& Labels)
{
	Labels.clear();
	for(int i=0;i<(int)(mLabels.size());i++)
	{
		Labels.push_back(mLabels[i]);
	}
}

void DestinData::GetShiftedImage(int ImageIndex, int RowShift, int ColShift, float** &fData )
{
    int R=4;
    int C=4;
    int size = mRows*mCols*sizeof(float);
    cout << "Single image is: " << size << " Bytes." << endl;
    cout << "Single image is: " << 256*256*sizeof(float) << " Bytes." << endl;
    //allocate memory if necessary...
    if ( fData==NULL )
    {
        fData = new float*[mRows];
        for (int rr=0;rr<mRows;rr++)
        {
            fData[rr]=new float[mCols];
        }
    }

    if ( ImageIndex!=mLastImageIndex )
    {
        //Load the image into the 50x50 buffer with the "0,0" offset...
        for(int r=0;r<50;r++)
        {
            for(int c=0;c<50;c++)
            {
                mImageWithOffset[r][c]=0;
            }
        }
        float** fImage = mImagePointer[ImageIndex];
        for(int r=0;r<mRows;r++)
        {
            for(int c=0;c<mCols;c++)
            {
                mImageWithOffset[r+R][c+C]=fImage[r][c];
            }
        }
    }

    //Now load the data using the offset provided...
    for(int r=0;r<mRows;r++)
    {
        for(int c=0;c<mCols;c++)
        {
            fData[r][c]=mImageWithOffset[r+RowShift][c+ColShift];
        }
    }
    mLastImageIndex=ImageIndex;
}

void DestinData::GetSubImage(int ImageIndex,int RowShift,int ColShift,
		int rS, int rE,int cS,int cE,float** &fSubImage)
{
	GetShiftedImage(ImageIndex,RowShift,ColShift,mShiftedImageCache);
	int nRows=rE-rS+1;
	int nCols=cE-cS+1;
	float* fRowOut;
	float* fShiftedImageCacheRow;
	for(int r=0;r<nRows;r++)
	{
		fRowOut = fSubImage[r];
		fShiftedImageCacheRow=mShiftedImageCache[r+rS];
		for(int c=0;c<nCols;c++)
		{
			*(fRowOut+c)=fShiftedImageCacheRow[c+cS];	
		}
	}
}

//note: this only does a 4x4 FFT and returns all 16 coefficients as magnitude in fSubImage
void DestinData::GetSubImageFFT(int ImageIndex,int RowShift,int ColShift,
        int rS, int rE,int cS,int cE,float** &fSubImage, float* fUniqueVector)
{
    GetSubImage(ImageIndex,RowShift,ColShift,rS,rE,cS,cE,fSubImage);
    DoSpecial4x4FFT(fSubImage,fUniqueVector);
}

void DestinData::GetSubImageVector(int ImageIndex,int RowShift,int ColShift,
		int rS, int rE,int cS,int cE,float** &fSubImage, float* fVector)
{
	GetShiftedImage(ImageIndex,RowShift,ColShift,mShiftedImageCache);
	int nRows=rE-rS+1;
	int nCols=cE-cS+1;
	float* fRowOut;
	float* fShiftedImageCacheRow;
	int kj=0;
	for(int r=0;r<nRows;r++)
	{
		fRowOut = fSubImage[r];
		fShiftedImageCacheRow=mShiftedImageCache[r+rS];
		for(int c=0;c<nCols;c++)
		{
			*(fRowOut+c)=fShiftedImageCacheRow[c+cS];	
			*(fVector+kj)=fShiftedImageCacheRow[c+cS];
			kj++;
		}
	}
}

void DestinData::WriteToCSV(int ImageIndex, int RowShift, int ColShift, char* cFile)
{
	float** fData = NULL;
	GetShiftedImage(ImageIndex, RowShift, ColShift, fData );
	std::ofstream stmCSV;
	stmCSV.open(cFile,ios::out);
	float* fRow;
	for(int r=0;r<mRows;r++)
	{
		fRow = fData[r];
		for(int c=0;c<mCols;c++)
		{
			stmCSV << *(fRow+c);
			if ( c != mCols-1 )
			{
				stmCSV << ",";
			}
		}
		stmCSV << endl;
	}
	stmCSV.close();
}

void DestinData::GetUniqueLabels(vector<int>& vLabels)
{
	vLabels.clear();
	vector<int>::iterator it = this->mUniqueLabels.begin();
	while ( it != mUniqueLabels.end() )
	{
		vLabels.push_back(*it++);
	}

}

void DestinData::GetIndicesForThisLabel(int iLabel, vector<int>& IndicesForThisLabel )
{
	IndicesForThisLabel.clear();

	map<int, vector<int> >::iterator it;

	it = mMapLabelToIndexVector.find(iLabel);
	if ( it!=mMapLabelToIndexVector.end() ) //find it
	{
		vector<int>::iterator vit;
		vit = it->second.begin();
		while ( vit != it->second.end() )
		{
			IndicesForThisLabel.push_back( *vit++ );
		}
	}
}

void DestinData::DoSpecial4x4FFT(float** &fSubImage, float* fUniqueVector)
{
	// I don't generally recommend doing FFTs like this, but since we only want 4x4 I didn't want to 
	// go to the trouble of adding an external library.  Plus you can do 4x4 with only adds & subtracts...
	int r,c;
	float fV;
	//Initialize the imaginary parts that we won't be 'hitting'...
	mIP[0][0]=0;mIP[2][0]=0;mIP[0][2]=0;mIP[2][2]=0;
	////////////////Row 0 Col 0///////
	fV = fSubImage[0][0];
	mRP[0][0]=fV; mRP[0][0]=fV; mRP[1][0]=fV; mRP[1][0]=fV; mRP[2][0]=fV; mRP[2][0]=fV; mRP[3][0]=fV; mRP[3][0]=fV; 
	mRP[0][1]=fV; mRP[0][1]=fV; mRP[1][1]=fV; mRP[1][1]=fV; mRP[2][1]=fV; mRP[2][1]=fV; mRP[3][1]=fV; mRP[3][1]=fV; 
	mRP[0][2]=fV; mRP[0][2]=fV; mRP[1][2]=fV; mRP[1][2]=fV; mRP[2][2]=fV; mRP[2][2]=fV; mRP[3][2]=fV; mRP[3][2]=fV; 
	mRP[0][3]=fV; mRP[0][3]=fV; mRP[1][3]=fV; mRP[1][3]=fV; mRP[2][3]=fV; mRP[2][3]=fV; mRP[3][3]=fV; mRP[3][3]=fV; 
	////////////////Row 0 Col 1///////
	fV = fSubImage[0][1];
	mRP[0][0]=mRP[0][0]+fV; mRP[1][0]=mRP[1][0]+fV; mRP[2][0]=mRP[2][0]+fV; mRP[3][0]=mRP[3][0]+fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[1][2]=mRP[1][2]-fV; mRP[2][2]=mRP[2][2]-fV; mRP[3][2]=mRP[3][2]-fV; 
	mIP[0][1]=-fV; mIP[0][1]=-fV; mIP[1][1]=-fV; mIP[1][1]=-fV; mIP[2][1]=-fV; mIP[2][1]=-fV; mIP[3][1]=-fV; mIP[3][1]=-fV; 
	mIP[0][3]=fV; mIP[0][3]=fV; mIP[1][3]=fV; mIP[1][3]=fV; mIP[2][3]=fV; mIP[2][3]=fV; mIP[3][3]=fV; mIP[3][3]=fV; 
	////////////////Row 0 Col 2///////
	fV = fSubImage[0][2];
	mRP[0][0]=mRP[0][0]+fV; mRP[1][0]=mRP[1][0]+fV; mRP[2][0]=mRP[2][0]+fV; mRP[3][0]=mRP[3][0]+fV; 
	mRP[0][1]=mRP[0][1]-fV; mRP[1][1]=mRP[1][1]-fV; mRP[2][1]=mRP[2][1]-fV; mRP[3][1]=mRP[3][1]-fV; 
	mRP[0][2]=mRP[0][2]+fV; mRP[1][2]=mRP[1][2]+fV; mRP[2][2]=mRP[2][2]+fV; mRP[3][2]=mRP[3][2]+fV; 
	mRP[0][3]=mRP[0][3]-fV; mRP[1][3]=mRP[1][3]-fV; mRP[2][3]=mRP[2][3]-fV; mRP[3][3]=mRP[3][3]-fV; 
	////////////////Row 0 Col 3///////
	fV = fSubImage[0][3];
	mRP[0][0]=mRP[0][0]+fV; mRP[1][0]=mRP[1][0]+fV; mRP[2][0]=mRP[2][0]+fV; mRP[3][0]=mRP[3][0]+fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[1][2]=mRP[1][2]-fV; mRP[2][2]=mRP[2][2]-fV; mRP[3][2]=mRP[3][2]-fV; 
	mIP[0][1]=mIP[0][1]+fV; mIP[1][1]=mIP[1][1]+fV; mIP[2][1]=mIP[2][1]+fV; mIP[3][1]=mIP[3][1]+fV; 
	mIP[0][3]=mIP[0][3]-fV; mIP[1][3]=mIP[1][3]-fV; mIP[2][3]=mIP[2][3]-fV; mIP[3][3]=mIP[3][3]-fV; 
	////////////////Row 1 Col 0///////
	fV = fSubImage[1][0];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[0][1]=mRP[0][1]+fV; mRP[2][1]=mRP[2][1]-fV; 
	mRP[0][2]=mRP[0][2]+fV; mRP[2][2]=mRP[2][2]-fV; mRP[0][3]=mRP[0][3]+fV; mRP[2][3]=mRP[2][3]-fV; 
	mIP[1][0]=-fV; mIP[1][0]=-fV; mIP[3][0]=fV; mIP[3][0]=fV; mIP[1][1]=mIP[1][1]-fV; mIP[3][1]=mIP[3][1]+fV; 
	mIP[1][2]=-fV; mIP[1][2]=-fV; mIP[3][2]=fV; mIP[3][2]=fV; mIP[1][3]=mIP[1][3]-fV; mIP[3][3]=mIP[3][3]+fV; 
	////////////////Row 1 Col 1///////
	fV = fSubImage[1][1];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[1][1]=mRP[1][1]-fV; mRP[3][1]=mRP[3][1]+fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[2][2]=mRP[2][2]+fV; mRP[1][3]=mRP[1][3]+fV; mRP[3][3]=mRP[3][3]-fV; 
	mIP[1][0]=mIP[1][0]-fV; mIP[3][0]=mIP[3][0]+fV; mIP[0][1]=mIP[0][1]-fV; mIP[2][1]=mIP[2][1]+fV; 
	mIP[1][2]=mIP[1][2]+fV; mIP[3][2]=mIP[3][2]-fV; mIP[0][3]=mIP[0][3]+fV; mIP[2][3]=mIP[2][3]-fV; 
	////////////////Row 1 Col 2///////
	fV = fSubImage[1][2];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[0][1]=mRP[0][1]-fV; mRP[2][1]=mRP[2][1]+fV; 
	mRP[0][2]=mRP[0][2]+fV; mRP[2][2]=mRP[2][2]-fV; mRP[0][3]=mRP[0][3]-fV; mRP[2][3]=mRP[2][3]+fV; 
	mIP[1][0]=mIP[1][0]-fV; mIP[3][0]=mIP[3][0]+fV; mIP[1][1]=mIP[1][1]+fV; mIP[3][1]=mIP[3][1]-fV; 
	mIP[1][2]=mIP[1][2]-fV; mIP[3][2]=mIP[3][2]+fV; mIP[1][3]=mIP[1][3]+fV; mIP[3][3]=mIP[3][3]-fV; 
	////////////////Row 1 Col 3///////
	fV = fSubImage[1][3];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[1][1]=mRP[1][1]+fV; mRP[3][1]=mRP[3][1]-fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[2][2]=mRP[2][2]+fV; mRP[1][3]=mRP[1][3]-fV; mRP[3][3]=mRP[3][3]+fV; 
	mIP[1][0]=mIP[1][0]-fV; mIP[3][0]=mIP[3][0]+fV; mIP[0][1]=mIP[0][1]+fV; mIP[2][1]=mIP[2][1]-fV; 
	mIP[1][2]=mIP[1][2]+fV; mIP[3][2]=mIP[3][2]-fV; mIP[0][3]=mIP[0][3]-fV; mIP[2][3]=mIP[2][3]+fV; 
	////////////////Row 2 Col 0///////
	fV = fSubImage[2][0];
	mRP[0][0]=mRP[0][0]+fV; mRP[1][0]=mRP[1][0]-fV; mRP[2][0]=mRP[2][0]+fV; mRP[3][0]=mRP[3][0]-fV; 
	mRP[0][1]=mRP[0][1]+fV; mRP[1][1]=mRP[1][1]-fV; mRP[2][1]=mRP[2][1]+fV; mRP[3][1]=mRP[3][1]-fV; 
	mRP[0][2]=mRP[0][2]+fV; mRP[1][2]=mRP[1][2]-fV; mRP[2][2]=mRP[2][2]+fV; mRP[3][2]=mRP[3][2]-fV; 
	mRP[0][3]=mRP[0][3]+fV; mRP[1][3]=mRP[1][3]-fV; mRP[2][3]=mRP[2][3]+fV; mRP[3][3]=mRP[3][3]-fV; 
	////////////////Row 2 Col 1///////
	fV = fSubImage[2][1];
	mRP[0][0]=mRP[0][0]+fV; mRP[1][0]=mRP[1][0]-fV; mRP[2][0]=mRP[2][0]+fV; mRP[3][0]=mRP[3][0]-fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[1][2]=mRP[1][2]+fV; mRP[2][2]=mRP[2][2]-fV; mRP[3][2]=mRP[3][2]+fV; 
	mIP[0][1]=mIP[0][1]-fV; mIP[1][1]=mIP[1][1]+fV; mIP[2][1]=mIP[2][1]-fV; mIP[3][1]=mIP[3][1]+fV; 
	mIP[0][3]=mIP[0][3]+fV; mIP[1][3]=mIP[1][3]-fV; mIP[2][3]=mIP[2][3]+fV; mIP[3][3]=mIP[3][3]-fV; 
	////////////////Row 2 Col 2///////
	fV = fSubImage[2][2];
	mRP[0][0]=mRP[0][0]+fV; mRP[1][0]=mRP[1][0]-fV; mRP[2][0]=mRP[2][0]+fV; mRP[3][0]=mRP[3][0]-fV; 
	mRP[0][1]=mRP[0][1]-fV; mRP[1][1]=mRP[1][1]+fV; mRP[2][1]=mRP[2][1]-fV; mRP[3][1]=mRP[3][1]+fV; 
	mRP[0][2]=mRP[0][2]+fV; mRP[1][2]=mRP[1][2]-fV; mRP[2][2]=mRP[2][2]+fV; mRP[3][2]=mRP[3][2]-fV; 
	mRP[0][3]=mRP[0][3]-fV; mRP[1][3]=mRP[1][3]+fV; mRP[2][3]=mRP[2][3]-fV; mRP[3][3]=mRP[3][3]+fV; 
	////////////////Row 2 Col 3///////
	fV = fSubImage[2][3];
	mRP[0][0]=mRP[0][0]+fV; mRP[1][0]=mRP[1][0]-fV; mRP[2][0]=mRP[2][0]+fV; mRP[3][0]=mRP[3][0]-fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[1][2]=mRP[1][2]+fV; mRP[2][2]=mRP[2][2]-fV; mRP[3][2]=mRP[3][2]+fV; 
	mIP[0][1]=mIP[0][1]+fV; mIP[1][1]=mIP[1][1]-fV; mIP[2][1]=mIP[2][1]+fV; mIP[3][1]=mIP[3][1]-fV; 
	mIP[0][3]=mIP[0][3]-fV; mIP[1][3]=mIP[1][3]+fV; mIP[2][3]=mIP[2][3]-fV; mIP[3][3]=mIP[3][3]+fV; 
	////////////////Row 3 Col 0///////
	fV = fSubImage[3][0];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[0][1]=mRP[0][1]+fV; mRP[2][1]=mRP[2][1]-fV; 
	mRP[0][2]=mRP[0][2]+fV; mRP[2][2]=mRP[2][2]-fV; mRP[0][3]=mRP[0][3]+fV; mRP[2][3]=mRP[2][3]-fV; 
	mIP[1][0]=mIP[1][0]+fV; mIP[3][0]=mIP[3][0]-fV; mIP[1][1]=mIP[1][1]+fV; mIP[3][1]=mIP[3][1]-fV; 
	mIP[1][2]=mIP[1][2]+fV; mIP[3][2]=mIP[3][2]-fV; mIP[1][3]=mIP[1][3]+fV; mIP[3][3]=mIP[3][3]-fV; 
	////////////////Row 3 Col 1///////
	fV = fSubImage[3][1];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[1][1]=mRP[1][1]+fV; mRP[3][1]=mRP[3][1]-fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[2][2]=mRP[2][2]+fV; mRP[1][3]=mRP[1][3]-fV; mRP[3][3]=mRP[3][3]+fV; 
	mIP[1][0]=mIP[1][0]+fV; mIP[3][0]=mIP[3][0]-fV; mIP[0][1]=mIP[0][1]-fV; mIP[2][1]=mIP[2][1]+fV; 
	mIP[1][2]=mIP[1][2]-fV; mIP[3][2]=mIP[3][2]+fV; mIP[0][3]=mIP[0][3]+fV; mIP[2][3]=mIP[2][3]-fV; 
	////////////////Row 3 Col 2///////
	fV = fSubImage[3][2];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[0][1]=mRP[0][1]-fV; mRP[2][1]=mRP[2][1]+fV; 
	mRP[0][2]=mRP[0][2]+fV; mRP[2][2]=mRP[2][2]-fV; mRP[0][3]=mRP[0][3]-fV; mRP[2][3]=mRP[2][3]+fV; 
	mIP[1][0]=mIP[1][0]+fV; mIP[3][0]=mIP[3][0]-fV; mIP[1][1]=mIP[1][1]-fV; mIP[3][1]=mIP[3][1]+fV; 
	mIP[1][2]=mIP[1][2]+fV; mIP[3][2]=mIP[3][2]-fV; mIP[1][3]=mIP[1][3]-fV; mIP[3][3]=mIP[3][3]+fV; 
	////////////////Row 3 Col 3///////
	fV = fSubImage[3][3];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[1][1]=mRP[1][1]-fV; mRP[3][1]=mRP[3][1]+fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[2][2]=mRP[2][2]+fV; mRP[1][3]=mRP[1][3]+fV; mRP[3][3]=mRP[3][3]-fV; 
	mIP[1][0]=mIP[1][0]+fV; mIP[3][0]=mIP[3][0]-fV; mIP[0][1]=mIP[0][1]+fV; mIP[2][1]=mIP[2][1]-fV; 
	mIP[1][2]=mIP[1][2]-fV; mIP[3][2]=mIP[3][2]+fV; mIP[0][3]=mIP[0][3]-fV; mIP[2][3]=mIP[2][3]+fV;

	//Finally take the magnitude...
	for(r=0;r<4;r++)
	{
		for(c=0;c<4;c++)
		{
			fSubImage[r][c]=(float)pow((double)(mRP[r][c]*mRP[r][c]+mIP[r][c]*mIP[r][c]),(double)0.5);
		}
	}

	//Unique values are 10: (0,0)-(2,2) and also (3,1)
	float* p = fUniqueVector;
	*p=fSubImage[0][0]/10.0; p++;
	*p=fSubImage[1][0]/10.0; p++;
	*p=fSubImage[2][0]/10.0; p++;

	*p=fSubImage[0][1]/10.0; p++;
	*p=fSubImage[1][1]/10.0; p++;
	*p=fSubImage[2][1]/10.0; p++;

	*p=fSubImage[0][2]/10.0; p++;
	*p=fSubImage[1][2]/10.0; p++;
	*p=fSubImage[2][2]/10.0; p++;

	*p=fSubImage[3][1]/10.0;
}
