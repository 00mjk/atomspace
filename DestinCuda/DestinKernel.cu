#include "hip/hip_runtime.h"
#include "DestinKernel.h"

// C/C++ headers
#include <stdio.h>
#include <math.h>
#include <vector>
#include <exception>
#include <stdexcept>
// Cuda header
#include <hip/hip_runtime.h>

using namespace std;

DestinKernel::DestinKernel(void)
{
	mRows=0;
	mCols=0;
	mStates=0;
	hipGetDeviceCount(&mDevices);
}

DestinKernel::~DestinKernel(void)
{
}

void DestinKernel::Create( int Rows, int Cols, int States)
{
    mRows = Rows;
    mCols = Cols;
    mStates = States;
}

__global__ void Layer()
{

}
