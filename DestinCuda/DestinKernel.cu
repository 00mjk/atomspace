#include "hip/hip_runtime.h"
#include "DestinKernel.h"

// C/C++ headers
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
// Cuda header
#include <hip/hip_runtime.h>
#include <hiprand.h>

const int AmountThreads = 128;

using namespace std;

__global__ void CalculateDistance( int States, int InputDimensionlity, float *InputData, float *CentroidVectorData, float *CentroidDist, int *WinningCentroids, float *CentroidStarvation, float *Output );
__global__ void UpdateStarvation( int States, float StarvationCoefficient, int *WinningCentroids, float *CentroidStarvation );

DestinKernel::DestinKernel( void )
{
    mID=0;
	mRows=0;
	mCols=0;
	mStates=0;
	mInputDimensionlity=0;
	hipGetDeviceCount(&mDevices);
	cout << "Kernel created" << endl;
}

DestinKernel::~DestinKernel( void )
{
    free ( mCentroidsVectorData );
    hipFree( dCentroidsVectorData );
    free ( mCentroidsDistance );
    hipFree( dCentroidsDistance );
    free ( mCentroidStarvation );
    hipFree( dCentroidStarvation );
    free ( mWinningCentroids );
    hipFree( dWinningCentroids );
    free ( mNodeOutput );
    hipFree( dNodeOutput );
    cout << "Kernel destroyed" << endl;
}

void DestinKernel::Create( int ID, int Rows, int Cols, int States, int InputDimensionlity, hiprandGenerator_t gen)
{
    mID = ID;
    mRows = Rows;
    mCols = Cols;
    mStates = States;
    mInputDimensionlity = InputDimensionlity;

    mSTARVATION_COEFFICIENT = 1.0/((float)InputDimensionlity*(float)InputDimensionlity);
    if ( mSTARVATION_COEFFICIENT < 1.0/512.0 )
    {
        mSTARVATION_COEFFICIENT=1.0/512.0;
    }

    // Define the data sizes
    // Size of de nodes is rows times columns
    sizeOfNodes = mRows*mCols;
    // Size of the data of nodes is rows times columns times centroids
    sizeOfNodeData = sizeOfNodes*mStates;
    // Size of the layer with all vectors is rows times columns times centroids times input vector
    sizeOfLayerData = sizeOfNodeData*mInputDimensionlity;

    // Array full with all the winning centroids of each node
    mWinningCentroids = new int[sizeOfNodes];
    hipMalloc( (void**)&dWinningCentroids, sizeOfNodes*sizeof(int) );

    // Node data contain the distance to the observation of all centroids (It's is empty the first run)
    mCentroidsDistance = new float[sizeOfNodeData];
    hipMalloc( (void**)&dCentroidsDistance, sizeOfNodeData*sizeof(float) );

    // Starvation data for all centroids
    mCentroidStarvation = new float[sizeOfNodeData];
    hipMalloc( (void**)&dCentroidStarvation, sizeOfNodeData*sizeof(float) );
    for(int i=0;i<sizeOfNodeData;i++)
    {
        mCentroidStarvation[i]=1.0f;
    }
    // Copy the data from host to device
    hipMemcpy(dCentroidStarvation, mCentroidStarvation, sizeOfNodeData*sizeof(float), hipMemcpyHostToDevice);

    // Output for next layer
    mNodeOutput = new float[sizeOfNodeData];
    hipMalloc( (void**)&dNodeOutput, sizeOfNodeData*sizeof(float) );

    // The layer data is the one that hold all vectors for all centroids inside each layer
    mCentroidsVectorData = new float[sizeOfLayerData];
    hipMalloc( (void**)&dCentroidsVectorData, sizeOfLayerData*sizeof(float) );
    // This is to fill the dLayerData with all random numbers between 0.0 and 1.0
    hiprandGenerateUniform( gen, dCentroidsVectorData, sizeOfLayerData );
    // TODO: (Re)move debug line.
    // hipMemcpy ( mCentroidVectorData, dCentroidVectorData, sizeOfLayerData*sizeof(float), hipMemcpyDeviceToHost );
}

void DestinKernel::DoDestin( float *Input )
{
    cout << "Layer: " << mID << endl;
    // Threads is the amount of thread inside each. block
    dim3 threads( AmountThreads );
    // Grid is the amount of blocks inside a grid.
    dim3 grid( mCols, mRows );
    // Cause of the use of dynamic shared memory you have to tell the kernel how much shared memory space you need for each block.
    int sharedMem = (mInputDimensionlity+mInputDimensionlity+mStates+mStates+mStates)*sizeof(float);
    // The launch of the kernel itself with centroids(states), dimension, input data and the Data of the layer itself
    CalculateDistance<<<grid, threads, sharedMem>>>( mStates, mInputDimensionlity, Input, dCentroidsVectorData, dCentroidsDistance, dWinningCentroids, dCentroidStarvation, dNodeOutput );
    UpdateStarvation<<<grid, threads>>>( mStates, mSTARVATION_COEFFICIENT, dWinningCentroids, dCentroidStarvation );

    // TODO: move this debug information or make sure the data is ready for other parts of DeSTIN.
    hipMemcpy(mCentroidsDistance, dCentroidsDistance, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mCentroidStarvation, dCentroidStarvation, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mNodeOutput, dNodeOutput, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mWinningCentroids, dWinningCentroids, sizeOfNodes*sizeof(int), hipMemcpyDeviceToHost);

    for(int r=0;r<mRows;r++)
    {
        for(int c=0;c<mCols;c++)
        {
            cout << "Node: " << r*mCols+c << endl;
            cout << "Winning: " << mWinningCentroids[r*mCols+c] << endl;
            for(int s=0;s<mStates;s++)
            {
                cout << "Centroid: " << s << " : " << mCentroidsDistance[(c+r*mCols)*mStates+s];
                cout << " Starvation: " << mCentroidStarvation[(c+r*mCols)*mStates+s];
                cout << " OutPut: " << mNodeOutput[(c+r*mCols)*mStates+s];
                cout << endl;
            }
            cout << endl;
        }
    }
}

__global__ void CalculateDistance( int States, int InputDimensionlity, float *InputData, float *CentroidVectorData, float *CentroidDist, int *WinningCentroids, float *CentroidStarvation, float *Output )
{
    // This is how to declare a shared memory inside CUDA.
    extern __shared__ float shared[];
    float* input = (float*)&shared;
    float* distance = (float*)&input[InputDimensionlity];
    float* winner = (float*)&distance[InputDimensionlity];
    float* winnerId = (float*)&winner[States];
    float* tPOS = (float*)&winnerId[States];

    // We use many threads they need to know where they have to do there work.
    // tid (Thread ID) is the amount of threads inside a block its a fixed amount it can be changed by changing: AmountThreads.
    // Keep in mind that CUDA threads should be in steps of 32 (each warp takes 4 clock cycles where each cycle calculate 8 threads)
    int tid = threadIdx.x;
    // bid (Block ID) this keeps track in witch node we are working you can ask the grid the size of the blocks used in x or y and on a Fermi or higher even z
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    // make sure the input data is inside shared memory this we are going to compare the amount of centroids.
    while(tid < InputDimensionlity)
    {
        // Put input data for node inside shared memory
        input[tid] = InputData[tid + bid * InputDimensionlity];
        // A trick for when the dimension is bigger then the amount of threads
        tid += blockDim.x;
    }
    // all threads have to be here to be sure shared memory is filled with the input.
    __syncthreads();

    // calculation distance in massive thread style.
    // keep track of the centroid
    int centroid = 0;
    while (centroid<States)
    {
        // reset the tid
        tid = threadIdx.x;
        while(tid < InputDimensionlity)
        {
            // This temp will have for a short while the calculation of input - centroid for position tid (one cell of the vector)
            float temp = 0.0f;
            // distance to input = (input - centroid)*(input - centroid)
            // Small formula to get to the right working position: dimension*centroids*block+current centroid*dimension+thread
            temp = (input[tid] - CentroidVectorData[InputDimensionlity*States*bid+centroid*InputDimensionlity+tid]);
            distance[tid] = temp * temp;
            // A trick for when the dimension is bigger then the amount of threads
            tid += blockDim.x;
        }
        // all threads have to wait here so we know all distance have been calculated
        __syncthreads();

        // bite wise divide by 2 (should be faster the /2)
        int d = InputDimensionlity >> 1;
        // Cause DeSTIN don't work with numbers that are 2^? we have to check for odd numbers
        int dOld = d*2;
        // a sum reduction, This is a common trick on CUDA to add shared memory instead of striding true memory
        // You have to use half the memory each step and each thread will add itself to with the other half.
        while (d != 0)
        {
            // reset the tid
            tid = threadIdx.x;
            dOld = dOld - d*2;
            while(tid < d)
            {
                // the adding calculation
                distance[tid] += distance[tid + d];

                // special case in case of odd number (As long as this don't happen to often it won't effect speed)
                if (dOld == 1 && tid == d-1)
                {
                    distance[tid] += distance[tid + d + 1];
                }
                tid += blockDim.x;
            }
            // Sync moment before starting with next iteration of reduction.
            __syncthreads();

            dOld = d;
            d >>= 1;
        }

        // Write distance to Node Data
        tid = threadIdx.x;
        float dist = 0.0f;
        if(tid == 0)
        {
            // square root on sum of the (input - centroid)*(input - centroid)
            dist = (sqrt(distance[tid]))*CentroidStarvation[centroid+bid*States];
            // Index of centroid inside node
            winnerId[centroid] = centroid;
            // Fill shared memory with distance of each centroid
            winner[centroid] = dist;
            // POS calculation from original DeSTIN
            tPOS[centroid] = (float)(1.0/(1e-9+(double)dist));
            // For debugging or analyzing saving the Distance to the observation
            // (Remember that you should copy the data from the device to the host and store it then)
            CentroidDist[centroid+bid*States] = dist;
        }
        // go to next centroid inside the node (bid is taking care of the other node)
        centroid++;
    }
    __syncthreads();

    // Reduction trick again to find winning centroid
    // (Looks like merge sort only instead of sorting everything just move the winning centroid to position 0)
    // The sum of tPOS will be done also cause we need that one later
    int d = States >> 1;
    int dOld = d*2;
    while (d != 0)
    {
        tid = threadIdx.x;
        dOld = dOld - d*2;
        while(tid < d)
        {
            // Adding tPOS
            tPOS[tid] += tPOS[tid + d];
            if(winner[tid] > winner[tid + d])
            {
                // Move winning centroid to the beginning
                winnerId[tid] = winnerId[tid + d];
            }

            if (dOld == 1 && tid == d-1)
            {
                // special case of odd numbers
                tPOS[tid] += tPOS[tid + d + 1];
                if(winner[tid] > winner[tid + d + 1])
                {
                    winnerId[tid] = winnerId[tid + d + 1];
                }
            }
            tid += blockDim.x;
        }
        // Sync moment before starting with next iteration of reduction.
        __syncthreads();

        dOld = d;
        d >>= 1;
    }
    // Write the winning centroid into there position
    tid = threadIdx.x;
    if(tid == 0)
    {
        WinningCentroids[bid] = winnerId[tid];
    }
    while(tid < States)
    {
        // This is the POS for all centroids (It looks like this is the input for the next layer also)
        // The output is missing the advice of higher layer
        Output[tid+bid*States] = (float)(1.0/(1e-9+(double)CentroidDist[tid+bid*States]))/tPOS[0];
        tid += blockDim.x;
    }
}

__global__ void UpdateStarvation( int States, float StarvationCoefficient, int *WinningCentroids, float *CentroidStarvation )
{
    // This is the updating starvation fast and quick to update all the nodes and reset the winning centroid
    // According to DeSTIN paper: The winning centroid starvation gets reset while the others starve more
    // Aldo this is the simple version of it it might be changed in the further cause this make the network also forget what it learn
    // when it is looking at something else for a very long time (Short and Long term memory)
    // for tid and bid see CalculateDistance kernel.
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;
    while(tid < States)
    {
        // Let all centroid starve
        CentroidStarvation[tid+bid*States] = (1.0f-StarvationCoefficient)*CentroidStarvation[tid+bid*States];
        // Reset winning centroid
        CentroidStarvation[WinningCentroids[bid]+bid*States] = 1.0f;
        tid += blockDim.x;
    }
}
