#include "hip/hip_runtime.h"
#include "DestinKernel.h"

// C/C++ headers
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
// Cuda header
#include <hip/hip_runtime.h>
#include <hiprand.h>

const int AmountThreads = 128;

using namespace std;

__global__ void Destin( int States, int InputDimensionlity, float *dInputData, float *dLayerData, float *dNodeData );
__global__ void sum(int a, int b, int *c);

DestinKernel::DestinKernel( void )
{
    mID=0;
	mRows=0;
	mCols=0;
	mStates=0;
	mInputDimensionlity=0;
	hipGetDeviceCount(&mDevices);
	cout << "Kernel created" << endl;
}

DestinKernel::~DestinKernel( void )
{
    free ( mLayerData ) ;
    hipFree( dLayerData );
    free ( mNodeData ) ;
    hipFree( dNodeData );
    cout << "Kernel destroyed" << endl;
}

void DestinKernel::Create( int ID, int Rows, int Cols, int States, int InputDimensionlity)
{
    mID = ID;
    mRows = Rows;
    mCols = Cols;
    mStates = States;
    mInputDimensionlity = InputDimensionlity;

    // Data holder for whole layer including centroids
    // Size of data holder is rows times columns.
    // Cause it needs to hold amount of centroids (mStates) including its vector the whole structure is time centroids and InputDimensionlity.
    sizeOfLayerData = mRows*mCols*mStates*mInputDimensionlity;
    mLayerData = new float[sizeOfLayerData];
    hipMalloc( (void**)&dLayerData, sizeOfLayerData*sizeof(float) );

    // hiprandGenerator_t is a CUDA version of rand
    // This fills the whole memory block with number between 0.0 and 1.0
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    // TODO: Put seed code at the place of 1
    hiprandSetPseudoRandomGeneratorSeed( gen, mID );
    hiprandGenerateUniform( gen, dLayerData, sizeOfLayerData );

    // TODO: Remove debug line.
    hipMemcpy ( mLayerData, dLayerData, sizeOfLayerData*sizeof(float), hipMemcpyDeviceToHost );

    // The generator have to be destroyed after use.
    hiprandDestroyGenerator( gen );
    // Node data contain the distance to the observation (It's is empty the first run)
    sizeOfNodeData = mRows*mCols*mStates;
    mNodeData = new float[sizeOfNodeData];
    hipMalloc( (void**)&dNodeData, sizeOfNodeData*sizeof(float) );
}

void DestinKernel::DoDestin( float *Input )
{
    // Threads is the amount of thread inside each block
    dim3 threads( AmountThreads );
    // Grid is the amount of blocks inside a grid
    dim3 grid( mCols, mRows );
    // The launch of the kernel itself with centroids(states), dimension, input data and the Data of the layer itself

    Destin<<<grid, threads, (mInputDimensionlity+mStates)*sizeof(float)>>>( mStates, mInputDimensionlity, Input, dLayerData, dNodeData );

    hipMemcpy(mNodeData, dNodeData, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    for(int r=0;r<mRows;r++)
    {
        for(int c=0;c<mCols;c++)
        {
            cout << "Node: " << r*mCols+c << endl;
            for(int s=0;s<mStates;s++)
            {
                cout << "Centroid: " << s << " : ";
                cout << mNodeData[r*mCols+c*mStates+s] << endl;
            }
            cout << endl;
        }
    }
}

__global__ void Destin( int States, int InputDimensionlity, float *dInputData, float *dLayerData, float *dNodeData )
{
    // This is how to declare a shared memory inside CUDA.
    extern __shared__ float shared[];
    float* input = (float*)&shared;
    float* distance = (float*)&input[InputDimensionlity*sizeof(float)];

    // We use many threads they need to know where they have to do there work.
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y*gridDim.x;

    // make sure the input data is inside shared memory this we are going to compare the amount of centroids.
    while(tid < InputDimensionlity)
    {
        // Put input data for node inside shared memory
        input[tid] = dInputData[tid+bid*InputDimensionlity];
        // A trick for when the dimension is bigger then the amount of threads
        tid += blockDim.x;
    }
    // all threads have to be here to be sure shared memory is filled with the input.
    __syncthreads();

    // calculation distance in massive thread style.
    int node = 0;
    while (node<States)
    {
        tid = threadIdx.x;
        while(tid < InputDimensionlity)
        {
            // Small formula to get to the right working position: dimension*centroids*block+current centroid*dimension+thread
            distance[tid] = input[tid] - dLayerData[InputDimensionlity*States*bid+node*InputDimensionlity+tid];
            distance[tid] = distance[tid] * distance[tid];
            // A trick for when the dimension is bigger then the amount of threads
            tid += blockDim.x;
        }
        // all threads have to wait here so we know all distance have been calculated
        __syncthreads();

        // bit wise divide by 2
        int d = InputDimensionlity >> 1;
        int dOld = d*2;
        tid = threadIdx.x;
        // a sum reduction
        while (d != 0)
        {
            dOld = dOld - d*2;
            while(tid < d)
            {
                distance[tid] += distance[tid + d];
                if (dOld == 1 && tid == d)
                {
                    distance[tid] += distance[tid+d+1];
                }
                tid += blockDim.x;
            }
            // Sync moment before starting with next iteration of reduction.
            __syncthreads();

            dOld = d;
            d >>= 1;
        }
        // Write distance to Node Data
        if(tid == 0)
        {
            dNodeData[node+bid*States] = sqrt(distance[tid]);
        }
        node++;
    }
}
