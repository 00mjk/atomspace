#include "hip/hip_runtime.h"
#include "DestinKernel.h"

// C/C++ headers
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
// Cuda header
#include <hip/hip_runtime.h>
#include <hiprand.h>

using namespace std;

__global__ void Destin( int States, float *image, float *dLayerData );
__global__ void DestinLast( int States, float *image, float *dLayerData, int inputRows, int inputColms );

DestinKernel::DestinKernel( void )
{
    mID=0;
	mRows=0;
	mCols=0;
	mStates=0;
	mInputDimensionlity=0;
	mLast = false;
	hipGetDeviceCount(&mDevices);
	cout << "Layer created" << endl;
}

DestinKernel::~DestinKernel( void )
{
    free ( mLayerData ) ;
    hipFree( dLayerData );
    cout << "Layer destroyed" << endl;
}

void DestinKernel::Create( int ID, int Rows, int Cols, int States, int InputDimensionlity, bool Last)
{
    mID = ID;
    mRows = Rows;
    mCols = Cols;
    mStates = States;
    mInputDimensionlity = InputDimensionlity;
    mLast = Last;

    // Data holder for whole layer including centroids
    // Size of data holder is rows times columns.
    // Cause it needs to hold amount of centroids (mStates) including its vector the whole structre is time centroids and InputDimensionlity.
    int size = mRows*mCols*mStates*mInputDimensionlity;
    mLayerData = (float*) calloc(size, sizeof(float) );
    hipMalloc( (void**)&dLayerData, size*sizeof(float) );

    // hiprandGenerator_t is a CUDA version of rand
    // This fills the whole memory block with number between 0.0 and 1.0
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    // TODO: Put seed code at the place of 1
    hiprandSetPseudoRandomGeneratorSeed( gen, 1 );
    hiprandGenerateUniform( gen, dLayerData, size );

    // TODO: Remove debug line.
    hipMemcpy ( mLayerData ,dLayerData , size*sizeof(float), hipMemcpyDeviceToHost );

    // The generator have to be destroyed after use.
    hiprandDestroyGenerator( gen );
}

void DestinKernel::DoDestin( float *image )
{
    dim3 threads( mInputDimensionlity);
    dim3 grid( mCols, mRows );
    if(mLast)
    {
        DestinLast<<<grid,threads>>>( mStates, image, dLayerData, 4, 4 );
    }
    else
    {
        Destin<<<grid,threads>>>( mStates, image, dLayerData );
    }
}

__global__ void DestinLast( int States, float *image, float *dLayerData, int inputRows, int inputColms )
{
    __shared__ float* observation;
    int x,y;
    x = blockDim.x;
    y = blockDim.y;
    threadIdx.x;
    threadIdx.y;
}

__global__ void Destin( int States, float *dInputLayerData, float *dOutLayerData )
{
    __shared__ float* observation;
    int x,y;
    x = blockDim.x;
    y = blockDim.y;
}
