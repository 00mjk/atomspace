#include "hip/hip_runtime.h"
#include "DestinKernel.h"

// C/C++ headers
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
// Cuda header
#include <hip/hip_runtime.h>
#include <hiprand.h>

using namespace std;

__global__ void Destin( int States, int InputDimensionlity, float *image, float *dLayerData );

DestinKernel::DestinKernel( void )
{
	mRows=0;
	mCols=0;
	mStates=0;
	mInputDimensionlity=0;
	hipGetDeviceCount(&mDevices);
	cout << "Layer created" << endl;
}

DestinKernel::~DestinKernel( void )
{
    free ( mLayerData ) ;
    hipFree( dLayerData );
    cout << "Layer destroyed" << endl;
}

void DestinKernel::Create( int Rows, int Cols, int States, int InputDimensionlity )
{
    mRows = Rows;
    mCols = Cols;
    mStates = States;
    mInputDimensionlity = InputDimensionlity;

    // Data holder for whole layer including centroids
    // Size of data holder is rows times columns.
    // Cause it needs to hold amount of centroids (mStates) including its vector the whole structre is time centroids and InputDimensionlity.
    int size = mRows*mCols*mStates*mInputDimensionlity;
    mLayerData = (float*) calloc(size, sizeof(float) );
    hipMalloc( (void**)&dLayerData, size*sizeof(float) );

    // hiprandGenerator_t is a CUDA version of rand
    // This fills the whole memory block with number between 0.0 and 1.0
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    // TODO: Put seed code at the place of 1
    hiprandSetPseudoRandomGeneratorSeed( gen, 1 );
    hiprandGenerateUniform( gen, dLayerData, size );
    // TODO: Remove debug line.
    hipMemcpy ( mLayerData ,dLayerData , size*sizeof(float), hipMemcpyDeviceToHost );
    hiprandDestroyGenerator( gen );
}

void DestinKernel::DoDestin( float *image )
{
    dim3 threads( 64, 1 );
    dim3 grid( mCols, mRows );
    Destin<<<grid,threads>>>( mStates, mInputDimensionlity, image, dLayerData );
}

__global__ void Destin( int States, int InputDimensionlity, float *image, float *dLayerData )
{
    __shared__ float* observation;
    int x,y;
    x = blockDim.x;
    y = blockDim.y;

}
