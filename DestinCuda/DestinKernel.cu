#include "hip/hip_runtime.h"
#include "DestinKernel.h"

// C/C++ headers
#include <stdio.h>
#include <math.h>
#include <vector>
#include <exception>
#include <stdexcept>
// Cuda header
#include <hip/hip_runtime.h>

using namespace std;

__global__ void Layer( int States, int InputDimensionlity, float *image, float *dLayerData );

DestinKernel::DestinKernel( void )
{
	mRows=0;
	mCols=0;
	mStates=0;
	mInputDimensionlity=0;
	hipGetDeviceCount(&mDevices);
}

DestinKernel::~DestinKernel( void )
{
    hipFree(dLayerData);
}

void DestinKernel::Create( int Rows, int Cols, int States, int InputDimensionlity )
{
    mRows = Rows;
    mCols = Cols;
    mStates = States;
    mInputDimensionlity = InputDimensionlity;
}

void DestinKernel::DoDestin( float *image )
{
    int size = mRows*mCols*mStates;
    hipMalloc( (void**)&dLayerData, size*sizeof(float) );

    dim3 threads(64, 1);
    dim3 grid(mCols, mRows);
    Layer<<<threads,grid>>>(mStates, mInputDimensionlity, image, dLayerData);
}

__global__ void Layer( int States, int InputDimensionlity, float *image, float *dLayerData )
{
    __shared__ float* observation;
    threadIdx.x;
    blockIdx.x;
}
