#include "hip/hip_runtime.h"
#include "DestinKernel.h"

// C/C++ headers
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
// Cuda header
#include <hip/hip_runtime.h>
#include <hiprand.h>

const int AmountThreads = 128;

using namespace std;

__global__ void CalculateDistance( int States, int InputDimensionlity, float *InputData, float *CentroidVectorData, float *CentroidData, int *WinningCentroids, float *CentroidStarvation );
__global__ void UpdateStarvation( int States, float StarvationCoefficient, int *WinningCentroids, float *CentroidStarvation );

DestinKernel::DestinKernel( void )
{
    mID=0;
	mRows=0;
	mCols=0;
	mStates=0;
	mInputDimensionlity=0;
	hipGetDeviceCount(&mDevices);
	cout << "Kernel created" << endl;
}

DestinKernel::~DestinKernel( void )
{
    free ( mCentroidVectorData ) ;
    hipFree( dCentroidVectorData );
    free ( mCentroidData ) ;
    hipFree( dCentroidData );
    free ( mCentroidStarvation ) ;
    hipFree( dCentroidStarvation );
    free ( mWinningCentroids ) ;
    hipFree( dWinningCentroids );
    cout << "Kernel destroyed" << endl;
}

void DestinKernel::Create( int ID, int Rows, int Cols, int States, int InputDimensionlity, hiprandGenerator_t gen)
{
    mID = ID;
    mRows = Rows;
    mCols = Cols;
    mStates = States;
    mInputDimensionlity = InputDimensionlity;

    mSTARVATION_COEFFICIENT = 1.0/((float)InputDimensionlity*(float)InputDimensionlity);
    if ( mSTARVATION_COEFFICIENT < 1.0/512.0 )
    {
        mSTARVATION_COEFFICIENT=1.0/512.0;
    }

    // Define the data sizes
    // Size of de nodes is rows times columns
    sizeOfNodes = mRows*mCols;
    // Size of the data of nodes is rows times columns times centroids
    sizeOfNodeData = sizeOfNodes*mStates;
    // Size of the layer with all vectors is rows times columns times centroids times input vector
    sizeOfLayerData = sizeOfNodeData*mInputDimensionlity;

    // Array full with all the winning centroids of each node
    mWinningCentroids = new int[sizeOfNodes];
    hipMalloc( (void**)&dWinningCentroids, sizeOfNodes*sizeof(int) );

    // Node data contain the distance to the observation of all centroids (It's is empty the first run)
    mCentroidData = new float[sizeOfNodeData];
    hipMalloc( (void**)&dCentroidData, sizeOfNodeData*sizeof(float) );

    // Starvation data for all centroids
    mCentroidStarvation = new float[sizeOfNodeData];
    hipMalloc( (void**)&dCentroidStarvation, sizeOfNodeData*sizeof(float) );
    for(int i=0;i<sizeOfNodeData;i++)
    {
        mCentroidStarvation[i]=1.0f;
    }
    hipMemcpy(dCentroidStarvation, mCentroidStarvation, sizeOfNodeData*sizeof(float), hipMemcpyHostToDevice);

    // The layer data is the one that hold all vectors for all centroids inside each layer
    mCentroidVectorData = new float[sizeOfLayerData];
    hipMalloc( (void**)&dCentroidVectorData, sizeOfLayerData*sizeof(float) );
    // This is to fill the dLayerData with all random numbers between 0.0 and 1.0
    hiprandGenerateUniform( gen, dCentroidVectorData, sizeOfLayerData );
    // TODO: (Re)move debug line.
    // hipMemcpy ( mCentroidVectorData, dCentroidVectorData, sizeOfLayerData*sizeof(float), hipMemcpyDeviceToHost );
}

void DestinKernel::DoDestin( float *Input )
{
    // Threads is the amount of thread inside each. block
    dim3 threads( AmountThreads );
    // Grid is the amount of blocks inside a grid.
    dim3 grid( mCols, mRows );
    // Cause of the use of dynamic shared memory you have to tell the kernel how much shared memory space you need for each block.
    int sharedMem = (mInputDimensionlity+mInputDimensionlity+mStates+mStates)*sizeof(float);
    // The launch of the kernel itself with centroids(states), dimension, input data and the Data of the layer itself
    CalculateDistance<<<grid, threads, sharedMem>>>( mStates, mInputDimensionlity, Input, dCentroidVectorData, dCentroidData, dWinningCentroids, dCentroidStarvation );
    cout << "Meuk: " << mSTARVATION_COEFFICIENT << endl;
    UpdateStarvation<<<grid, threads>>>( mStates, mSTARVATION_COEFFICIENT, dWinningCentroids, dCentroidStarvation );
    hipMemcpy(mCentroidData, dCentroidData, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mCentroidStarvation, dCentroidStarvation, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mWinningCentroids, dWinningCentroids, sizeOfNodes*sizeof(int), hipMemcpyDeviceToHost);
    for(int r=0;r<mRows;r++)
    {
        for(int c=0;c<mCols;c++)
        {
            cout << "Node: " << r*mCols+c << endl;
            cout << "Winning: " << mWinningCentroids[r*mCols+c] << endl;
            for(int s=0;s<mStates;s++)
            {
                cout << "Centroid: " << s << " : ";
                cout << mCentroidData[(c+r*mCols)*mStates+s];
                cout << " Starvation: " << mCentroidStarvation[(c+r*mCols)*mStates+s];
                cout << endl;
            }
            cout << endl;
        }
    }
}

__global__ void CalculateDistance( int States, int InputDimensionlity, float *InputData, float *CentroidVectorData, float *CentroidData, int *WinningCentroids, float *CentroidStarvation )
{
    // This is how to declare a shared memory inside CUDA.
    extern __shared__ float shared[];
    float* input = (float*)&shared;
    float* distance = (float*)&input[InputDimensionlity];
    float* winner = (float*)&distance[InputDimensionlity];
    float* winnerId = (float*)&winner[States];

    // We use many threads they need to know where they have to do there work.
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    // make sure the input data is inside shared memory this we are going to compare the amount of centroids.
    while(tid < InputDimensionlity)
    {
        // Put input data for node inside shared memory
        input[tid] = InputData[tid + bid * InputDimensionlity];
        // A trick for when the dimension is bigger then the amount of threads
        tid += blockDim.x;
    }
    // all threads have to be here to be sure shared memory is filled with the input.
    __syncthreads();

    // calculation distance in massive thread style.
    int node = 0;
    float temp = 0.0f;
    while (node<States)
    {
        tid = threadIdx.x;
        while(tid < InputDimensionlity)
        {
            // Small formula to get to the right working position: dimension*centroids*block+current centroid*dimension+thread
            temp = (input[tid] - CentroidVectorData[InputDimensionlity*States*bid+node*InputDimensionlity+tid]);
            distance[tid] = temp * temp;
            // A trick for when the dimension is bigger then the amount of threads
            tid += blockDim.x;
        }
        // all threads have to wait here so we know all distance have been calculated
        __syncthreads();

        // bit wise divide by 2
        int d = InputDimensionlity >> 1;
        int dOld = d*2;
        // a sum reduction
        while (d != 0)
        {
            tid = threadIdx.x;
            dOld = dOld - d*2;
            while(tid < d)
            {
                distance[tid] += distance[tid + d];

                if (dOld == 1 && tid == d-1)
                {
                    distance[tid] += distance[tid + d + 1];
                }
                tid += blockDim.x;
            }
            // Sync moment before starting with next iteration of reduction.
            __syncthreads();

            dOld = d;
            d >>= 1;
        }

        // Write distance to Node Data
        tid = threadIdx.x;
        float dist = 0.0f;
        if(tid == 0)
        {
            dist = (sqrt(distance[tid]))*CentroidStarvation[node+bid*States];
            winner[node] = dist;
            winnerId[node] = node;
            CentroidData[node+bid*States] = dist;
        }
        node++;
    }
    __syncthreads();

    int d = States >> 1;
    int dOld = d*2;
    // Reduction trick to find winning centroid
    while (d != 0)
    {
        tid = threadIdx.x;
        dOld = dOld - d*2;
        while(tid < d)
        {
            if(winner[tid] > winner[tid + d])
            {
                winnerId[tid] = winnerId[tid + d];
            }

            if (dOld == 1 && tid == d-1)
            {
                if(winner[tid] > winner[tid + d + 1])
                {
                    winnerId[tid] = winnerId[tid + d + 1];
                }
            }
            tid += blockDim.x;
        }
        // Sync moment before starting with next iteration of reduction.
        __syncthreads();

        dOld = d;
        d >>= 1;
    }
    // Write the winning centroid into there position
    tid = threadIdx.x;
    if(tid == 0)
    {
        WinningCentroids[bid] = winnerId[tid];
    }
}

__global__ void UpdateStarvation( int States, float StarvationCoefficient, int *WinningCentroids, float *CentroidStarvation )
{
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;
    while(tid < States)
    {
        CentroidStarvation[tid+bid*States] = (1.0f-StarvationCoefficient)*CentroidStarvation[tid+bid*States];
        CentroidStarvation[WinningCentroids[bid]+bid*States] = 1.0f;
        tid += blockDim.x;
    }
}
