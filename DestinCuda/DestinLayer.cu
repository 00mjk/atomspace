#include "hip/hip_runtime.h"
#include "DestinLayer.h"

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <math.h>
#include <vector>
#include <stdexcept>

using namespace std;
using std::tr1::shared_ptr;

DestinLayer::DestinLayer(void)
{
	mRows=0;
	mCols=0;
	mGTLabel=-1;	//ground truth label, i.e, "letter A", "digit B", "Dog"
	mSignalIndex=-1; //index, for example if we have 100 things in the data set this could be 0-99
	mMovementNumber=-1; // which movement we are on
	mObservationNumber=-1; //which observation we are working on iteratively
    mMaxChildrenPerNode=0; //after AssignChildrenAndParents(int,int,int,int) is called, this stores
                            //the maximum number of children any of these nodes has.
}

DestinLayer::~DestinLayer(void)
{
	ClearAndDestroy();
}

void DestinLayer::ClearAndDestroy(void)
{
//	for(int r=0;r<(int)(mDestinNodeUnits.size());r++)
//	{
//		delete mDestinNodeUnits[r];
//	}
//	mDestinNodeUnits.clear();
}

//This assumes that the csv files are in this directory and have a convention:
// Layer01_Row_03_Col_01_CENTROIDS.csv
// Layer01_Row_03_Col_01_SMAX.csv
void DestinLayer::OverrideCentroidsAndSMaxWithCSVFiles(int Layer, char* sDirectory)
{
	char cCentroidFileName[128];
	char cSMAXFileName[128];

	int r,c;
	for(r=0;r<mRows;r++)
	{
		for(c=0;c<mCols;c++)
		{
			sprintf(cCentroidFileName,"%s/Layer%.2d_Row_%.2d_Col_%.2d_CENTROIDS.csv",sDirectory,Layer,r,c);			
			sprintf(cSMAXFileName,"%s/Layer%.2d_Row_%.2d_Col_%.2d_SMAX.csv",sDirectory,Layer,r,c);	
//			mDestinNodeUnits[c+r*mCols]->OverrideCentroidsWithCSV(cCentroidFileName);
//			mDestinNodeUnits[c+r*mCols]->OverrideSMaxWithCSV(cSMAXFileName);
		}
	}
}

//Write to a stream...
bool DestinLayer::WriteToStream(std::ofstream& stmOutput)
{
	stmOutput.write( (char*)&mRows, sizeof(mRows) );
	stmOutput.write( (char*)&mCols, sizeof(mCols) );

	stmOutput.write( (char*)&mGTLabel, sizeof(mGTLabel) );
	stmOutput.write( (char*)&mSignalIndex, sizeof(mSignalIndex) );
	stmOutput.write( (char*)&mMovementNumber, sizeof(mMovementNumber) );
	stmOutput.write( (char*)&mObservationNumber, sizeof(mObservationNumber) );

//	DestinNode* MyNode;
//	int r,c;
//	for(r=0;r<mRows;r++)
//	{
//		for(c=0;c<mCols;c++)
//		{
//			MyNode = this->GetPointerToNode(r,c);
//			MyNode->WriteToStream(stmOutput);
//		}
//	}

	return true;
}

// Read from a stream...
bool DestinLayer::ReadFromStream(std::ifstream& stmInput)
{
	stmInput.read( (char*)&mRows, sizeof(mRows) );
	stmInput.read( (char*)&mCols, sizeof(mCols) );

	stmInput.read( (char*)&mGTLabel, sizeof(mGTLabel) );
	stmInput.read( (char*)&mSignalIndex, sizeof(mSignalIndex) );
	stmInput.read( (char*)&mMovementNumber, sizeof(mMovementNumber) );
	stmInput.read( (char*)&mObservationNumber, sizeof(mObservationNumber) );

//	DestinNode* MyNode;
//	int r,c;
//	for (r=0;r<mDestinNodeUnits.size();r++)
//	{
//		delete mDestinNodeUnits[r];
//	}
//	mDestinNodeUnits.clear();
//	for(r=0;r<mRows;r++)
//	{
//		for(c=0;c<mCols;c++)
//		{
//			MyNode = new DestinNode();
//			MyNode->ReadFromStream(stmInput);
//			mDestinNodeUnits.push_back(MyNode);
//		}
//	}

	return true;
}


bool DestinLayer::operator == (DestinLayer& o)
{
	bool bReturn = true;
	if ( mRows != o.mRows ) bReturn = false;
	if ( mCols != o.mCols ) bReturn = false;
	if ( mGTLabel != o.mGTLabel ) bReturn = false;
	if ( mSignalIndex != o.mSignalIndex ) bReturn = false;
	if ( mMovementNumber != o.mMovementNumber ) bReturn = false;
	if ( mObservationNumber != o.mObservationNumber ) bReturn = false;

//	DestinNode* MyNode;
//	DestinNode* ItsNode;
//
//	if ( bReturn )
//	{
//		int r,c;
//		for(r=0;r<mRows;r++)
//		{
//			for(c=0;c<mCols;c++)
//			{
//				MyNode = this->GetPointerToNode(r,c);
//				ItsNode = o.GetPointerToNode(r,c);
//				if ( !(*MyNode==*ItsNode) )
//				{
//					bool bTest = (*MyNode==*ItsNode);
//					bReturn = false;
//				}
//			}
//		}
//	}


	return bReturn;
}

void DestinLayer::Create( int Rows, int Cols, int States, int ParentStates, int InputDimensionality, int inputDimensionalities[],
                          int DType, bool bBinaryPOS, bool bAveraging, bool bUseStarvationCoefficient, int PSSAUpdateDelay,
                          bool bIgnoreAdvice, double dcMu, double dcSigma, double dcRho, bool bUseDecayingLearningRate,
                          int iDecayKickInPoint, float fRhoThresholdPoint, bool bUseRhoDerivative, bool bConstrainInitialCentroids,
                          int iBlocksToProcess,	int LayerNumber, int iMovementsForCluster, bool BasicOnlineClustering,
                          float FixedRate, bool bDoGoodPOS, int SequenceLength, bool bTopNode )
{
    DestinLayer();
	mRows=Rows;
	mCols=Cols;
//	mDestinNodeUnits.clear();
//	DestinNode* p;
//
//	int iNodeID;
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			if(inputDimensionalities!=NULL){
//				InputDimensionality = inputDimensionalities[r*mCols + c];
//			}
//			iNodeID = c+1000*r+LayerNumber*10000;
//			p=new DestinNode();
//			p->Create(States,ParentStates,InputDimensionality,
//				DType,
//				bBinaryPOS,
//				bAveraging,
//				bUseStarvationCoefficient,
//				bIgnoreAdvice,
//				dcMu, dcSigma, dcRho,
//				bUseDecayingLearningRate,
//				iDecayKickInPoint,
//				fRhoThresholdPoint,
//				bUseRhoDerivative,
//				bConstrainInitialCentroids,
//				iBlocksToProcess,
//				iNodeID,
//				iMovementsForCluster,
//				BasicOnlineClustering,
//				FixedRate,r,c,LayerNumber, bDoGoodPOS, SequenceLength,bTopNode);
//
//			p->SetPSSAUpdateDelay(PSSAUpdateDelay);
//			mDestinNodeUnits.push_back(p);
//
//		}
//	}
}
void DestinLayerLatch::SetDiagnosticData(int GTLabel, int SignalIndex, int MovementNumber, int ObservationNumber)
{
	mGTLabel=GTLabel;
	mSignalIndex=SignalIndex;
	mMovementNumber=MovementNumber;
	mObservationNumber=ObservationNumber;

}

void DestinLayer::SetDiagnosticData(int GTLabel, int SignalIndex, int MovementNumber, int ObservationNumber)
{
	mGTLabel=GTLabel;
	mSignalIndex=SignalIndex;
	mMovementNumber=MovementNumber;
	mObservationNumber=ObservationNumber;
}

void DestinLayer::ClearValidOutputFlag()
{
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			mDestinNodeUnits[c+r*mCols]->ClearValidOutputFlag(); // put relevant data in latch...
//		}
//	}
}

void DestinLayer::SetNextUpdateCountAndSequenceLength(int UC, int SL)
{
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			mDestinNodeUnits[c+r*mCols]->SetNextUpdateCountAndSequenceLength(UC,SL);
//		}
//	}
}

void DestinLayer::SetCompileCentroidShiftMetrics(bool b)
{
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			mDestinNodeUnits[c+r*mCols]->SetCompileCentroidShiftMetrics(b);
//		}
//	}
}

void DestinLayer::LatchData(DestinLayerLatch& oLatch)
{
//	DestinNodeLatchData* LatchData;
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			LatchData=oLatch.GetPointerToNode(r,c); //get pointer to nodes latch data...
//			mDestinNodeUnits[c+r*mCols]->GetDataForLatching(*LatchData); // put relevant data in latch...
//			// For feedback, you can use LatchData
//		}
//	}
//	//set the diagnostic data too...
//	oLatch.SetDiagnosticData(mGTLabel,mSignalIndex,mMovementNumber,mObservationNumber);
}


//DestinNode* DestinLayer::GetPointerToNode(int r, int c)
//{
//	return mDestinNodeUnits[c+r*mCols];
//}

DestinLayerLatch::DestinLayerLatch(void)
{
	mRows=0;
	mCols=0;
	mGTLabel=-1;	//ground truth label, i.e, "letter A", "digit B", "Dog"
	mSignalIndex=-1; //index, for example if we have 100 things in the data set this could be 0-99
	mMovementNumber=-1; // which movement we are on
	mObservationNumber=-1; //which observation we are working on iteratively

}

DestinLayerLatch::~DestinLayerLatch(void)
{
//	for(int r=0;r<(int)(mDestinNodeLatchUnits.size());r++)
//	{
//		delete mDestinNodeLatchUnits[r];
//	}
}
//DestinNodeLatchData* DestinLayerLatch::GetPointerToNode(int r, int c)
//{
//	return mDestinNodeLatchUnits[c+r*mCols];
//}

void DestinLayerLatch::Create(int Rows, int Cols, int States )
{
    DestinLayerLatch();
	mRows=Rows;
	mCols=Cols;
	//mDestinNodeLatchUnits.clear();

//	DestinNodeLatchData* p;// = mDestinNodeLatchUnits;
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			p=new DestinNodeLatchData();
//			p->Create(States);
//			mDestinNodeLatchUnits.push_back(p);
//		}
//	}
}

DestinLayer::FamilySizes DestinLayer::calcFamilySizes(int parentLayerNodeCount, int childNodeCount) {
	const int nn = parentLayerNodeCount;
	FamilySizes fs;

	fs.nLargeFamilies = 0;
	fs.smallFamilySize = 0;
	fs.largeFamilySize = 0;
	fs.nSmallFamilies = 0;
	if (parentLayerNodeCount != 0) {
		fs.nLargeFamilies = childNodeCount % nn;
		fs.smallFamilySize = childNodeCount / nn;
		fs.largeFamilySize = fs.smallFamilySize + 1;
		fs.nSmallFamilies = nn - fs.nLargeFamilies;

		if(fs.nLargeFamilies == 0){
			//if there are only small familyies, make them the big families,
			//so thay there are always big families
			fs.nLargeFamilies = fs.nSmallFamilies;
			fs.largeFamilySize = fs.smallFamilySize;
			fs.smallFamilySize = 0;
			fs.nSmallFamilies = 0;
		}
	}
	return fs;
}

void DestinLayer::AssignChildrenAndParents(int childLayerRows, int childLayerCols, int parentLayerRows, int parentLayerCols){
    const int nParentNodes = parentLayerRows * parentLayerCols;
    const int nChildNodes = childLayerCols * childLayerRows;
    //const int nn=mDestinNodeUnits.size();//number of nodes
    const int nn=0;//number of nodes

    if(nn<1){
        throw logic_error("Can't assign children and parents for a layer with no nodes.");
    }

    if(nChildNodes!=0 && nn > nChildNodes){
        throw invalid_argument("This layer can't have more nodes than its child layer.");
    }

    if(nParentNodes!=0 && nParentNodes > nn ){
        throw invalid_argument("Parent layer can't have more nodes than this layer.");
    }

    int r, c;

	FamilySizes fs = calcFamilySizes(nParentNodes,nn);

    //assign parent nodes
    for(int pn = 0 ; pn < fs.nLargeFamilies ; pn ++){
        for(int cn = 0 ; cn  < fs.largeFamilySize ; cn ++){
            r = pn / parentLayerCols;
            c = pn % parentLayerCols;
            //mDestinNodeUnits.at(pn * fs.largeFamilySize + cn)->AddParentNode(r,c);
        }
    }
    int offset  = fs.nLargeFamilies * fs.largeFamilySize;
    for(int pn = 0 ; pn < fs.nSmallFamilies ; pn ++){
        for(int cn = 0 ; cn < fs.smallFamilySize ; cn++){
            r = (pn + fs.nLargeFamilies) / parentLayerCols;
            c = (pn + fs.nLargeFamilies) % parentLayerCols;
            //mDestinNodeUnits.at(pn * fs.smallFamilySize + cn + offset)->AddParentNode(r,c);
        }
    }

	fs = calcFamilySizes(nn,nChildNodes);

	//assign child nodes
	
    for(int pn = 0 ; pn < fs.nLargeFamilies ; pn++){
        for(int cn = 0 ; cn < fs.largeFamilySize ; cn++){
            r = (pn * fs.largeFamilySize + cn) / childLayerCols;
            c = (pn * fs.largeFamilySize + cn) % childLayerCols;
            //mDestinNodeUnits.at(pn )->AddChildNode(r,c);
        }
    }
    offset  = fs.nLargeFamilies * fs.largeFamilySize;

    for(int pn = 0 ; pn < fs.nSmallFamilies ; pn++){
        for(int cn = 0 ; cn< fs.smallFamilySize ; cn++){
            int index = pn * fs.smallFamilySize + cn + offset;
            r = index / childLayerCols;
            c = index % childLayerCols;
            //mDestinNodeUnits.at(pn + fs.nLargeFamilies)->AddChildNode(r,c);
        }
    }
    mMaxChildrenPerNode = fs.largeFamilySize;
	return;
}

//This assumes layer=0 is the sensory interface layer...with no children...
// ...and layer NumberOfLayers-1 is the top layer with no parents
//uses the 'default' 4:1 method UNLESS bUsesTransformationalLayer is true in which cases layers 1-up are 4:1 but 0-1 are 1:1
void DestinLayer::AssignChildrenAndParents(int Layer, int NumberOfLayers, bool bUsesTransformationalLayer) 
{
	int parentR;
	int parentC;

	if ( bUsesTransformationalLayer )
	{
		//Assign the children of each node here assuming a 4x1 relationship, i.e., I have 4 children laid out in 2-d space...
		// except the first two layers which are 1:1
		if ( Layer==1 )
		{
			for (int r=0;r<mRows;r++)
			{
				for(int c=0;c<mCols;c++)
				{
					//mDestinNodeUnits[c+r*mCols]->AddChildNode(r,c);
				}	//c
			}	//r
		}
		else if ( Layer >=2 )
		{
			for (int r=0;r<mRows;r++)
			{
				for(int c=0;c<mCols;c++)
				{
					// to match MATLAB, add the child nodes by row first...
					for (int ChildCol=2*c;ChildCol<=2*c+1;ChildCol++)
					{
						for (int ChildRow=2*r;ChildRow<=2*r+1;ChildRow++)
						{
							//The children here are ChildRow,ChildCol...
							//mDestinNodeUnits[c+r*mCols]->AddChildNode(ChildRow,ChildCol);
						}	//ChildCol
					}	//ChildRow
				}	//c
			}	//r
		}	//Layer

		//Now assign the parent! This is a little odder...

		if ( Layer != NumberOfLayers-1 )
		{
			if ( Layer==0 )
			{
				for (int r=0;r<mRows;r++)
				{
					for(int c=0;c<mCols;c++)
					{
						//The parents here are r c since this is 1:1
						//mDestinNodeUnits[c+r*mCols]->AddParentNode(r,c);
					}	//c
				}	//r	
			}
			else
			{
				for (int r=0;r<mRows;r++)
				{
					for(int c=0;c<mCols;c++)
					{
						//The parents here are floor(mRows/2),floor(mCols/2)
						parentR=(int)floor((float)r/(float)2.0);
						parentC=(int)floor((float)c/(float)2.0);
						//mDestinNodeUnits[c+r*mCols]->AddParentNode(parentR,parentC);
					}	//c
				}	//r		
			}
		}
	}
	else
	{
		//Assign the children of each node here assuming a 4x1 relationship, i.e., I have 4 children laid out in 2-d space...
		if ( Layer != 0 )
		{
			for (int r=0;r<mRows;r++)
			{
				for(int c=0;c<mCols;c++)
				{
					// to match MATLAB, add the child nodes by row first...
					for (int ChildCol=2*c;ChildCol<=2*c+1;ChildCol++)
					{
						for (int ChildRow=2*r;ChildRow<=2*r+1;ChildRow++)
						{
							//The children here are ChildRow,ChildCol...
							//mDestinNodeUnits[c+r*mCols]->AddChildNode(ChildRow,ChildCol);
						}	//ChildCol
					}	//ChildRow

				}	//c
			}	//r
		}	//Layer

		//Now assign the parent! This is a little odder...

		if ( Layer != NumberOfLayers-1 )
		{
			for (int r=0;r<mRows;r++)
			{
				for(int c=0;c<mCols;c++)
				{
					//The parents here are floor(mRows/2),floor(mCols/2)
					parentR=(int)floor((float)r/(float)2.0);
					parentC=(int)floor((float)c/(float)2.0);
					//mDestinNodeUnits[c+r*mCols]->AddParentNode(parentR,parentC);
				}	//c
			}	//r		
		}
	}

}

