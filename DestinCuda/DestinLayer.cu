#include "hip/hip_runtime.h"
#include "DestinLayer.h"

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <math.h>
#include <vector>
#include <stdexcept>

using namespace std;

DestinLayer::DestinLayer(void)
{
	mRows=0;
	mCols=0;
	mGTLabel=-1;	//ground truth label, i.e, "letter A", "digit B", "Dog"
	mSignalIndex=-1; //index, for example if we have 100 things in the data set this could be 0-99
	mMovementNumber=-1; // which movement we are on
	mObservationNumber=-1; //which observation we are working on iteratively
    mMaxChildrenPerNode=0; //after AssignChildrenAndParents(int,int,int,int) is called, this stores
                            //the maximum number of children any of these nodes has.
}

DestinLayer::~DestinLayer(void)
{
	ClearAndDestroy();
}

void DestinLayer::ClearAndDestroy(void)
{
//	for(int r=0;r<(int)(mDestinNodeUnits.size());r++)
//	{
//		delete mDestinNodeUnits[r];
//	}
//	mDestinNodeUnits.clear();
}

void DestinLayer::Create( int Rows, int Cols, int States, int ParentStates, int InputDimensionality, int inputDimensionalities[],
                          int DType, bool bBinaryPOS, bool bAveraging, bool bUseStarvationCoefficient, int PSSAUpdateDelay,
                          bool bIgnoreAdvice, double dcMu, double dcSigma, double dcRho, bool bUseDecayingLearningRate,
                          int iDecayKickInPoint, float fRhoThresholdPoint, bool bUseRhoDerivative, bool bConstrainInitialCentroids,
                          int iBlocksToProcess,	int LayerNumber, int iMovementsForCluster, bool BasicOnlineClustering,
                          float FixedRate, bool bDoGoodPOS, int SequenceLength, bool bTopNode )
{
    DestinLayer();
	mRows=Rows;
	mCols=Cols;
//	mDestinNodeUnits.clear();
//	DestinNode* p;
//
//	int iNodeID;
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			if(inputDimensionalities!=NULL){
//				InputDimensionality = inputDimensionalities[r*mCols + c];
//			}
//			iNodeID = c+1000*r+10000*LayerNumber;
//			p=new DestinNode();
//			p->Create(States,ParentStates,InputDimensionality,
//				DType,
//				bBinaryPOS,
//				bAveraging,
//				bUseStarvationCoefficient,
//				bIgnoreAdvice,
//				dcMu, dcSigma, dcRho,
//				bUseDecayingLearningRate,
//				iDecayKickInPoint,
//				fRhoThresholdPoint,
//				bUseRhoDerivative,
//				bConstrainInitialCentroids,
//				iBlocksToProcess,
//				iNodeID,
//				iMovementsForCluster,
//				BasicOnlineClustering,
//				FixedRate,r,c,LayerNumber, bDoGoodPOS, SequenceLength,bTopNode);
//
//			p->SetPSSAUpdateDelay(PSSAUpdateDelay);
//			mDestinNodeUnits.push_back(p);
//
//		}
//	}
}

//This assumes layer=0 is the sensory interface layer...with no children...
// ...and layer NumberOfLayers-1 is the top layer with no parents
//uses the 'default' 4:1 method UNLESS bUsesTransformationalLayer is true in which cases layers 1-up are 4:1 but 0-1 are 1:1
void DestinLayer::AssignChildrenAndParents(int Layer, int NumberOfLayers, bool bUsesTransformationalLayer)
{
    int parentR;
    int parentC;

    if ( bUsesTransformationalLayer )
    {
        //Assign the children of each node here assuming a 4x1 relationship, i.e., I have 4 children laid out in 2-d space...
        // except the first two layers which are 1:1
        if ( Layer==1 )
        {
            for (int r=0;r<mRows;r++)
            {
                for(int c=0;c<mCols;c++)
                {
                    //mDestinNodeUnits[c+r*mCols]->AddChildNode(r,c);
                }   //c
            }   //r
        }
        else if ( Layer >=2 )
        {
            for (int r=0;r<mRows;r++)
            {
                for(int c=0;c<mCols;c++)
                {
                    // to match MATLAB, add the child nodes by row first...
                    for (int ChildCol=2*c;ChildCol<=2*c+1;ChildCol++)
                    {
                        for (int ChildRow=2*r;ChildRow<=2*r+1;ChildRow++)
                        {
                            //The children here are ChildRow,ChildCol...
                            //mDestinNodeUnits[c+r*mCols]->AddChildNode(ChildRow,ChildCol);
                        }   //ChildCol
                    }   //ChildRow
                }   //c
            }   //r
        }   //Layer

        //Now assign the parent! This is a little odder...

        if ( Layer != NumberOfLayers-1 )
        {
            if ( Layer==0 )
            {
                for (int r=0;r<mRows;r++)
                {
                    for(int c=0;c<mCols;c++)
                    {
                        //The parents here are r c since this is 1:1
                        //mDestinNodeUnits[c+r*mCols]->AddParentNode(r,c);
                    }   //c
                }   //r
            }
            else
            {
                for (int r=0;r<mRows;r++)
                {
                    for(int c=0;c<mCols;c++)
                    {
                        //The parents here are floor(mRows/2),floor(mCols/2)
                        parentR=(int)floor((float)r/(float)2.0);
                        parentC=(int)floor((float)c/(float)2.0);
                        //mDestinNodeUnits[c+r*mCols]->AddParentNode(parentR,parentC);
                    }   //c
                }   //r
            }
        }
    }
    else
    {
        //Assign the children of each node here assuming a 4x1 relationship, i.e., I have 4 children laid out in 2-d space...
        if ( Layer != 0 )
        {
            for (int r=0;r<mRows;r++)
            {
                for(int c=0;c<mCols;c++)
                {
                    // to match MATLAB, add the child nodes by row first...
                    for (int ChildCol=2*c;ChildCol<=2*c+1;ChildCol++)
                    {
                        for (int ChildRow=2*r;ChildRow<=2*r+1;ChildRow++)
                        {
                            //The children here are ChildRow,ChildCol...
                            //mDestinNodeUnits[c+r*mCols]->AddChildNode(ChildRow,ChildCol);
                        }   //ChildCol
                    }   //ChildRow

                }   //c
            }   //r
        }   //Layer

        //Now assign the parent! This is a little odder...

        if ( Layer != NumberOfLayers-1 )
        {
            for (int r=0;r<mRows;r++)
            {
                for(int c=0;c<mCols;c++)
                {
                    //The parents here are floor(mRows/2),floor(mCols/2)
                    parentR=(int)floor((float)r/(float)2.0);
                    parentC=(int)floor((float)c/(float)2.0);
                    //mDestinNodeUnits[c+r*mCols]->AddParentNode(parentR,parentC);
                }   //c
            }   //r
        }
    }
}

DestinLayerLatch::DestinLayerLatch(void)
{
    mRows=0;
    mCols=0;
    mGTLabel=-1;    //ground truth label, i.e, "letter A", "digit B", "Dog"
    mSignalIndex=-1; //index, for example if we have 100 things in the data set this could be 0-99
    mMovementNumber=-1; // which movement we are on
    mObservationNumber=-1; //which observation we are working on iteratively

}

DestinLayerLatch::~DestinLayerLatch(void)
{
//  for(int r=0;r<(int)(mDestinNodeLatchUnits.size());r++)
//  {
//      delete mDestinNodeLatchUnits[r];
//  }
}
//DestinNodeLatchData* DestinLayerLatch::GetPointerToNode(int r, int c)
//{
//  return mDestinNodeLatchUnits[c+r*mCols];
//}

void DestinLayerLatch::Create(int Rows, int Cols, int States )
{
    DestinLayerLatch();
    mRows=Rows;
    mCols=Cols;
    //mDestinNodeLatchUnits.clear();

//  DestinNodeLatchData* p;// = mDestinNodeLatchUnits;
//  for(int r=0;r<mRows;r++)
//  {
//      for(int c=0;c<mCols;c++)
//      {
//          p=new DestinNodeLatchData();
//          p->Create(States);
//          mDestinNodeLatchUnits.push_back(p);
//      }
//  }
}
