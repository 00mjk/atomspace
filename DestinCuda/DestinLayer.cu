#include "hip/hip_runtime.h"
#include "DestinLayer.h"

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <math.h>
#include <vector>
#include <stdexcept>
#include <boost/tr1/memory.hpp>

using namespace std;
using std::tr1::shared_ptr;

DestinLayer::DestinLayer(void)
{
	mRows=0;
	mCols=0;
	mGTLabel=-1;	//ground truth label, i.e, "letter A", "digit B", "Dog"
	mSignalIndex=-1; //index, for example if we have 100 things in the data set this could be 0-99
	mMovementNumber=-1; // which movement we are on
	mObservationNumber=-1; //which observation we are working on iteratively
    mMaxChildrenPerNode=0; //after AssignChildrenAndParents(int,int,int,int) is called, this stores
                            //the maximum number of children any of these nodes has.
}

DestinLayer::~DestinLayer(void)
{
	ClearAndDestroy();
}

void DestinLayer::ClearAndDestroy(void)
{
//	for(int r=0;r<(int)(mDestinNodeUnits.size());r++)
//	{
//		delete mDestinNodeUnits[r];
//	}
//	mDestinNodeUnits.clear();
}

// This assumes that the csv files are in this directory and have a convention:
// Layer01_Row_03_Col_01_CENTROIDS.csv
// Layer01_Row_03_Col_01_SMAX.csv
void DestinLayer::OverrideCentroidsAndSMaxWithCSVFiles(int Layer, char* sDirectory)
{
	char cCentroidFileName[128];
	char cSMAXFileName[128];

	for(int r=0;r<mRows;r++)
	{
		for(int c=0;c<mCols;c++)
		{
			sprintf(cCentroidFileName,"%s/Layer%.2d_Row_%.2d_Col_%.2d_CENTROIDS.csv",sDirectory,Layer,r,c);			
			sprintf(cSMAXFileName,"%s/Layer%.2d_Row_%.2d_Col_%.2d_SMAX.csv",sDirectory,Layer,r,c);	
//			mDestinNodeUnits[c+r*mCols]->OverrideCentroidsWithCSV(cCentroidFileName);
//			mDestinNodeUnits[c+r*mCols]->OverrideSMaxWithCSV(cSMAXFileName);
		}
	}
}

// Write to a stream.
bool DestinLayer::WriteToStream(std::ofstream& stmOutput)
{
	stmOutput.write( (char*)&mRows, sizeof(mRows) );
	stmOutput.write( (char*)&mCols, sizeof(mCols) );
	stmOutput.write( (char*)&mGTLabel, sizeof(mGTLabel) );
	stmOutput.write( (char*)&mSignalIndex, sizeof(mSignalIndex) );
	stmOutput.write( (char*)&mMovementNumber, sizeof(mMovementNumber) );
	stmOutput.write( (char*)&mObservationNumber, sizeof(mObservationNumber) );

//	DestinNode* MyNode;
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			MyNode = this->GetPointerToNode(r,c);
//			MyNode->WriteToStream(stmOutput);
//		}
//	}

	return true;
}

// Read from a stream...
bool DestinLayer::ReadFromStream(std::ifstream& stmInput)
{
	stmInput.read( (char*)&mRows, sizeof(mRows) );
	stmInput.read( (char*)&mCols, sizeof(mCols) );

	stmInput.read( (char*)&mGTLabel, sizeof(mGTLabel) );
	stmInput.read( (char*)&mSignalIndex, sizeof(mSignalIndex) );
	stmInput.read( (char*)&mMovementNumber, sizeof(mMovementNumber) );
	stmInput.read( (char*)&mObservationNumber, sizeof(mObservationNumber) );

//	DestinNode* MyNode;
//	ClearAndDestroy();
//
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			MyNode = new DestinNode();
//			MyNode->ReadFromStream(stmInput);
//			mDestinNodeUnits.push_back(MyNode);
//		}
//	}

	return true;
}


bool DestinLayer::operator == (DestinLayer& o)
{
	bool bReturn = true;
	if ( mRows != o.mRows ) bReturn = false;
	if ( mCols != o.mCols ) bReturn = false;
	if ( mGTLabel != o.mGTLabel ) bReturn = false;
	if ( mSignalIndex != o.mSignalIndex ) bReturn = false;
	if ( mMovementNumber != o.mMovementNumber ) bReturn = false;
	if ( mObservationNumber != o.mObservationNumber ) bReturn = false;

//	DestinNode* MyNode;
//	DestinNode* ItsNode;
//
//	if ( bReturn )
//	{
//		for(int r=0;r<mRows;r++)
//		{
//			for(int c=0;c<mCols;c++)
//			{
//				MyNode = this->GetPointerToNode(r,c);
//				ItsNode = o.GetPointerToNode(r,c);
//				if ( !(*MyNode==*ItsNode) )
//				{
//					bool bTest = (*MyNode==*ItsNode);
//					bReturn = false;
//				}
//			}
//		}
//	}

	return bReturn;
}

void DestinLayer::Create( int Rows, int Cols, int States, int ParentStates, int InputDimensionality, int inputDimensionalities[],
                          int DType, bool bBinaryPOS, bool bAveraging, bool bUseStarvationCoefficient, int PSSAUpdateDelay,
                          bool bIgnoreAdvice, double dcMu, double dcSigma, double dcRho, bool bUseDecayingLearningRate,
                          int iDecayKickInPoint, float fRhoThresholdPoint, bool bUseRhoDerivative, bool bConstrainInitialCentroids,
                          int iBlocksToProcess,	int LayerNumber, int iMovementsForCluster, bool BasicOnlineClustering,
                          float FixedRate, bool bDoGoodPOS, int SequenceLength, bool bTopNode )
{
    DestinLayer();
	mRows=Rows;
	mCols=Cols;
//	mDestinNodeUnits.clear();
//	DestinNode* p;
//
//	int iNodeID;
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			if(inputDimensionalities!=NULL){
//				InputDimensionality = inputDimensionalities[r*mCols + c];
//			}
//			iNodeID = c+1000*r+10000*LayerNumber;
//			p=new DestinNode();
//			p->Create(States,ParentStates,InputDimensionality,
//				DType,
//				bBinaryPOS,
//				bAveraging,
//				bUseStarvationCoefficient,
//				bIgnoreAdvice,
//				dcMu, dcSigma, dcRho,
//				bUseDecayingLearningRate,
//				iDecayKickInPoint,
//				fRhoThresholdPoint,
//				bUseRhoDerivative,
//				bConstrainInitialCentroids,
//				iBlocksToProcess,
//				iNodeID,
//				iMovementsForCluster,
//				BasicOnlineClustering,
//				FixedRate,r,c,LayerNumber, bDoGoodPOS, SequenceLength,bTopNode);
//
//			p->SetPSSAUpdateDelay(PSSAUpdateDelay);
//			mDestinNodeUnits.push_back(p);
//
//		}
//	}
}
void DestinLayerLatch::SetDiagnosticData(int GTLabel, int SignalIndex, int MovementNumber, int ObservationNumber)
{
	mGTLabel=GTLabel;
	mSignalIndex=SignalIndex;
	mMovementNumber=MovementNumber;
	mObservationNumber=ObservationNumber;

}

void DestinLayer::SetDiagnosticData(int GTLabel, int SignalIndex, int MovementNumber, int ObservationNumber)
{
	mGTLabel=GTLabel;
	mSignalIndex=SignalIndex;
	mMovementNumber=MovementNumber;
	mObservationNumber=ObservationNumber;
}

void DestinLayer::ClearValidOutputFlag()
{
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			mDestinNodeUnits[c+r*mCols]->ClearValidOutputFlag(); // put relevant data in latch...
//		}
//	}
}

void DestinLayer::SetNextUpdateCountAndSequenceLength(int UC, int SL)
{
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			mDestinNodeUnits[c+r*mCols]->SetNextUpdateCountAndSequenceLength(UC,SL);
//		}
//	}
}

void DestinLayer::SetCompileCentroidShiftMetrics(bool b)
{
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			mDestinNodeUnits[c+r*mCols]->SetCompileCentroidShiftMetrics(b);
//		}
//	}
}

void DestinLayer::LatchData(DestinLayerLatch& oLatch)
{
//	DestinNodeLatchData* LatchData;
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			LatchData=oLatch.GetPointerToNode(r,c); //get pointer to nodes latch data...
//			mDestinNodeUnits[c+r*mCols]->GetDataForLatching(*LatchData); // put relevant data in latch...
//			// For feedback, you can use LatchData
//		}
//	}
//	//set the diagnostic data too...
//	oLatch.SetDiagnosticData(mGTLabel,mSignalIndex,mMovementNumber,mObservationNumber);
}


//DestinNode* DestinLayer::GetPointerToNode(int r, int c)
//{
//	return mDestinNodeUnits[c+r*mCols];
//}

DestinLayerLatch::DestinLayerLatch(void)
{
	mRows=0;
	mCols=0;
	mGTLabel=-1;	//ground truth label, i.e, "letter A", "digit B", "Dog"
	mSignalIndex=-1; //index, for example if we have 100 things in the data set this could be 0-99
	mMovementNumber=-1; // which movement we are on
	mObservationNumber=-1; //which observation we are working on iteratively

}

DestinLayerLatch::~DestinLayerLatch(void)
{
//	for(int r=0;r<(int)(mDestinNodeLatchUnits.size());r++)
//	{
//		delete mDestinNodeLatchUnits[r];
//	}
}
//DestinNodeLatchData* DestinLayerLatch::GetPointerToNode(int r, int c)
//{
//	return mDestinNodeLatchUnits[c+r*mCols];
//}

void DestinLayerLatch::Create(int Rows, int Cols, int States )
{
    DestinLayerLatch();
	mRows=Rows;
	mCols=Cols;
	//mDestinNodeLatchUnits.clear();

//	DestinNodeLatchData* p;// = mDestinNodeLatchUnits;
//	for(int r=0;r<mRows;r++)
//	{
//		for(int c=0;c<mCols;c++)
//		{
//			p=new DestinNodeLatchData();
//			p->Create(States);
//			mDestinNodeLatchUnits.push_back(p);
//		}
//	}
}

DestinLayer::FamilySizes DestinLayer::calcFamilySizes(int parentLayerNodeCount, int childNodeCount) {
	const int nn = parentLayerNodeCount;
	FamilySizes fs;

	fs.nLargeFamilies = 0;
	fs.smallFamilySize = 0;
	fs.largeFamilySize = 0;
	fs.nSmallFamilies = 0;
	if (parentLayerNodeCount != 0) {
		fs.nLargeFamilies = childNodeCount % nn;
		fs.smallFamilySize = childNodeCount / nn;
		fs.largeFamilySize = fs.smallFamilySize + 1;
		fs.nSmallFamilies = nn - fs.nLargeFamilies;

		if(fs.nLargeFamilies == 0){
			//if there are only small familyies, make them the big families,
			//so thay there are always big families
			fs.nLargeFamilies = fs.nSmallFamilies;
			fs.largeFamilySize = fs.smallFamilySize;
			fs.smallFamilySize = 0;
			fs.nSmallFamilies = 0;
		}
	}
	return fs;
}

//This assumes layer=0 is the sensory interface layer...with no children...
// ...and layer NumberOfLayers-1 is the top layer with no parents
//uses the 'default' 4:1 method UNLESS bUsesTransformationalLayer is true in which cases layers 1-up are 4:1 but 0-1 are 1:1
void DestinLayer::AssignChildrenAndParents(int Layer, int NumberOfLayers, bool bUsesTransformationalLayer) 
{
	int parentR;
	int parentC;

	if ( bUsesTransformationalLayer )
	{
		//Assign the children of each node here assuming a 4x1 relationship, i.e., I have 4 children laid out in 2-d space...
		// except the first two layers which are 1:1
		if ( Layer==1 )
		{
			for (int r=0;r<mRows;r++)
			{
				for(int c=0;c<mCols;c++)
				{
					//mDestinNodeUnits[c+r*mCols]->AddChildNode(r,c);
				}	//c
			}	//r
		}
		else if ( Layer >=2 )
		{
			for (int r=0;r<mRows;r++)
			{
				for(int c=0;c<mCols;c++)
				{
					// to match MATLAB, add the child nodes by row first...
					for (int ChildCol=2*c;ChildCol<=2*c+1;ChildCol++)
					{
						for (int ChildRow=2*r;ChildRow<=2*r+1;ChildRow++)
						{
							//The children here are ChildRow,ChildCol...
							//mDestinNodeUnits[c+r*mCols]->AddChildNode(ChildRow,ChildCol);
						}	//ChildCol
					}	//ChildRow
				}	//c
			}	//r
		}	//Layer

		//Now assign the parent! This is a little odder...

		if ( Layer != NumberOfLayers-1 )
		{
			if ( Layer==0 )
			{
				for (int r=0;r<mRows;r++)
				{
					for(int c=0;c<mCols;c++)
					{
						//The parents here are r c since this is 1:1
						//mDestinNodeUnits[c+r*mCols]->AddParentNode(r,c);
					}	//c
				}	//r	
			}
			else
			{
				for (int r=0;r<mRows;r++)
				{
					for(int c=0;c<mCols;c++)
					{
						//The parents here are floor(mRows/2),floor(mCols/2)
						parentR=(int)floor((float)r/(float)2.0);
						parentC=(int)floor((float)c/(float)2.0);
						//mDestinNodeUnits[c+r*mCols]->AddParentNode(parentR,parentC);
					}	//c
				}	//r		
			}
		}
	}
	else
	{
		//Assign the children of each node here assuming a 4x1 relationship, i.e., I have 4 children laid out in 2-d space...
		if ( Layer != 0 )
		{
			for (int r=0;r<mRows;r++)
			{
				for(int c=0;c<mCols;c++)
				{
					// to match MATLAB, add the child nodes by row first...
					for (int ChildCol=2*c;ChildCol<=2*c+1;ChildCol++)
					{
						for (int ChildRow=2*r;ChildRow<=2*r+1;ChildRow++)
						{
							//The children here are ChildRow,ChildCol...
							//mDestinNodeUnits[c+r*mCols]->AddChildNode(ChildRow,ChildCol);
						}	//ChildCol
					}	//ChildRow

				}	//c
			}	//r
		}	//Layer

		//Now assign the parent! This is a little odder...

		if ( Layer != NumberOfLayers-1 )
		{
			for (int r=0;r<mRows;r++)
			{
				for(int c=0;c<mCols;c++)
				{
					//The parents here are floor(mRows/2),floor(mCols/2)
					parentR=(int)floor((float)r/(float)2.0);
					parentC=(int)floor((float)c/(float)2.0);
					//mDestinNodeUnits[c+r*mCols]->AddParentNode(parentR,parentC);
				}	//c
			}	//r		
		}
	}
}
